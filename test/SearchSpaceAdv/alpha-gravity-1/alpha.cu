#include "hip/hip_runtime.h"
{% include "./EdgeSortTest/beta.cu" %}
{% include "./MovelistTest/beta.cu" %}

__constant__ unsigned char c_ucFourPermutations[{{ fit_nr_four_permutations }}][{{ glob_nr_tile_orientations }}];

struct xFourPermutation {
	unsigned short PermIndex;
	unsigned short WalkIndex;
	__device__ xFourPermutation(unsigned short __usPermIndex);
	__device__ unsigned short ucWalk();
	__device__ bool bNotTraversed();
};

__device__ xFourPermutation::xFourPermutation(unsigned short __usPermIndex) {
    this->PermIndex = __usPermIndex % 24;
    this->WalkIndex = 0;
}

__device__ unsigned short xFourPermutation::ucWalk() {
    //Require c_ucFourPermutations to be numbers 1-4 (NOT 0-3)
    this->WalkIndex++;
    if (this->WalkIndex - 1 < mNrTileOrientations) {
        return c_ucFourPermutations[this->PermIndex][this->WalkIndex - 1] - 1;
    } else return 0;
}

__device__ bool xFourPermutation::bNotTraversed() {
    //Require c_ucFourPermutations to be numbers 1-4 (NOT 0-3)
    if (this->WalkIndex >= mNrTileOrientations) {
        return false;
    } else return true;
}

extern "C++"{
template<int Length>
struct xLinearIterator {
	unsigned short WalkIndex;
	__device__ xLinearIterator(unsigned short __usPermIndex);
	__device__ unsigned short ucWalk();
	__device__ bool bNotTraversed();
};

template<int Length>
__device__ xLinearIterator<Length>::xLinearIterator(unsigned short __usPermIndex) {
    //this->WalkIndex = 0;
}

template<int Length>
__device__ unsigned short xLinearIterator<Length>::ucWalk() {
    //Require c_fFourPermutations to be numbers 1-4 (NOT 0-3)
    this->WalkIndex++;
    if (this->WalkIndex - 1 < Length) {
        return this->WalkIndex - 1;
    } else return 0;
}

template<int Length>        
__device__ bool xLinearIterator<Length>::bNotTraversed() {
    //Require c_fFourPermutations to be numbers 1-4 (NOT 0-3)
    if (this->WalkIndex >= Length) {
        return false;
    } else return true;
}

struct xCell {
	unsigned char data;
        __device__ void set_Orient(unsigned char __uiOrient);
        __device__ void set_Type(unsigned char __uiType);
        __device__ unsigned char get_xType(void);
        __device__ unsigned char get_xOrient(void);
        __device__ unsigned char get_xCell(void);
        __device__ void set_xCell(unsigned char __ucVal);
};

__device__ void xCell::set_Orient(unsigned char __uiOrient) {
	__uiOrient = __uiOrient % mNrTileOrientations;
	//unsigned char DBGVAL1 = this->data & (255-3);
	//unsigned char DBGVAL2 = __uiOrient;
	//unsigned char DBGVAL3 = this->data & (255-3) + __uiOrient;
	//I THINK THIS FUNCTION DOES NOT WORK!
	this->data = ((this->data & (255-3) ) + __uiOrient);
}

__device__ void xCell::set_Type(unsigned char __uiType) {
#ifndef __NON_FERMI
	if (__uiType > 63) {
		printf("xCell: TileType exceeded 63 limit!\n");
	}
#endif
	this->data = (this->data & 3) + (__uiType << 2);
}

__device__ void xCell::set_xCell(unsigned char __ucVal) {
	this->data = __ucVal;
}

__device__ unsigned char xCell::get_xType(void) {
	return this->data >> 2;
}

__device__ unsigned char xCell::get_xOrient(void) {
	return (this->data & 3);
}

__device__ unsigned char xCell::get_xCell(void) {
	return this->data;
}

struct xCellGrid {
	union {
		xCell multi_d[m_fit_DimGridX][m_fit_DimGridY][m_fit_NrRedundancyGridDepth][mWarpSize];
		xCell mix_d[m_fit_DimGridX * m_fit_DimGridY][m_fit_NrRedundancyGridDepth][mWarpSize];
		xCell one_d[m_fit_DimGridX * m_fit_DimGridY * mWarpSize	* m_fit_NrRedundancyGridDepth];
	} data;

	__device__ void Initialise(xThreadInfo __xThreadInfo, unsigned char __red);
        __device__ xCell get_xCell(xThreadInfo __xThreadInfo, unsigned char __x, unsigned char __y, unsigned char __red);
        __device__ bool set_xCell(xThreadInfo __xThreadInfo, unsigned char __x, unsigned char __y, unsigned char __red, unsigned char __val);
        __device__ xCell xGetNeighbourCell(xThreadInfo __xThreadInfo, unsigned char __x, unsigned char __y, unsigned char __red, unsigned char __dir);
        __device__ uchar2 xGetNeighbourCellCoords(unsigned char __x, unsigned char __y, unsigned char __dir);
        __device__ bool xCompareRed(xThreadInfo __xThreadInfo, unsigned char __red);
        __device__ void print(xThreadInfo __xThreadInfo, xGenomeSet *__xGenomeSet);
};

__device__ void xCellGrid::Initialise(xThreadInfo __xThreadInfo,
		unsigned char __red) {
	//Surefire-version:
	/*for (int i = 0; i < m_fit_DimGridX; i++) {
		for (int j = 0; j < m_fit_DimGridY; j++) {
			this->data.multi_d[i][j][__red][__xThreadInfo.BankId()].set_xCell(mEMPTY_CELL);
		}
	}*/
        /*for (int i = 0; i < m_fit_DimGridX; i++) {
                for (int j = 0; j < m_fit_DimGridY; j++) {
                        this->data.multi_d[i*j][__red][__xThreadInfo.BankId()].set_xCell(mEMPTY_CELL);
                }
        }*/
        short offset = (m_fit_DimGridX*m_fit_DimGridY) % mBankSize;
        short myshare = (m_fit_DimGridX*m_fit_DimGridY - offset) / mBankSize; 
        //short one_d_off = m_fit_DimGridX*m_fit_DimGridY*m_fit_NrRedundancyAssemblies*__xThreadInfo.BankId() + m_fit_DimGridX*m_fit_DimGridY*__red; 
        //_fit_DimGridX*m_fit_DimGridY;

        for(int i=0;i<myshare;i++){
                this->data.mix_d[__xThreadInfo.WarpId()*myshare + i][__red][__xThreadInfo.BankId()].set_xCell(mEMPTY_CELL); 
        }
        if(__xThreadInfo.WarpId()==mBankSize-1){
                for(int i=0;i<offset;i++){
                        //this->data.one_d[one_d_off + mBankSize*myshare + i].set_xCell(mEMPTY_CELL);
                        this->data.mix_d[mBankSize*myshare + i][__red][__xThreadInfo.BankId()].set_xCell(mEMPTY_CELL);
                }
        }
        //__syncthreads(); 
}

__device__ xCell xCellGrid::get_xCell(xThreadInfo __xThreadInfo, unsigned char __x, unsigned char __y, unsigned char __red) {
	if ((__x < m_fit_DimGridX) && (__y < m_fit_DimGridY)) {
		return this->data.multi_d[__x][__y][__red][__xThreadInfo.BankId()];
	} else {
		xCell TmpCell;
		TmpCell.set_xCell(mEMPTY_CELL);
		return TmpCell;
	}
}

__device__ bool xCellGrid::set_xCell(xThreadInfo __xThreadInfo, unsigned char __x, unsigned char __y, unsigned char __red, unsigned char __val) {
	if ((__x < m_fit_DimGridX - 1) && (__y < m_fit_DimGridY - 1)) {
		this->data.multi_d[__x][__y][__red][__xThreadInfo.BankId()].set_xCell(
				__val);
		return true;
	} else if (__x == (m_fit_DimGridX - 1) || (__y == (m_fit_DimGridY - 1))) {
		//UnboundUND condition! Return false.
		this->data.multi_d[__x][__y][__red][__xThreadInfo.BankId()].set_xCell(
				__val);
		return false;
	} else {
		return false;
	}
}

__device__ xCell xCellGrid::xGetNeighbourCell(xThreadInfo __xThreadInfo, unsigned char __x, unsigned char __y, unsigned char __red, unsigned char __dir) {
	uchar2 TmpCoords = xGetNeighbourCellCoords(__x, __y, __dir);
	return this->get_xCell(__xThreadInfo, TmpCoords.x, TmpCoords.y, __red);
}

__device__ uchar2 xCellGrid::xGetNeighbourCellCoords(unsigned char __x, unsigned char __y, unsigned char __dir) {
	switch (__dir) {
	case 1: //EAST
		return make_uchar2(__x + 1, __y);
		//break;
	case 3: //WEST
		return make_uchar2(__x - 1, __y);
		//break;
	case 2: //SOUTH
		return make_uchar2(__x, __y + 1);
		//break;
	case 0: //NORTH
		return make_uchar2(__x, __y - 1);
		//break;
	default:
		break;
	}
	return make_uchar2(mEMPTY_CELL, mEMPTY_CELL);
}

__device__ bool xCellGrid::xCompareRed(xThreadInfo __xThreadInfo, unsigned char __red) {
        unsigned char TmpNextDir = (__red + 1) % m_fit_NrRedundancyGridDepth;
	unsigned char TmpIsDifferent = 0;
	for (int i = 0; i < m_fit_DimGridX * m_fit_DimGridY; i++) {
		if (this->data.mix_d[i][__red][__xThreadInfo.BankId()].get_xCell() != this->data.mix_d[i][TmpNextDir][__xThreadInfo.BankId()].get_xCell() ) {
		    TmpIsDifferent = 1;
		    break;
		}
	}
	if (!TmpIsDifferent)
		return true;
	else
		return false;
}

struct xFitnessGrid {
	texture<xCell, 2> *grid;
	__device__ unsigned char get_xCell(unsigned char i, unsigned char j);
};

struct xAssembly {
	struct {
		xCellGrid grid;
		xEdgeSort edgesort;
		xMoveList<uchar2> movelist;
		xAssemblyFlags flags[mWarpSize];
		hiprandState *states;//[mWarpSize];
		unsigned int synccounter[mWarpSize]; //Will be used to synchronize between Warps
                int2 gravity[mWarpSize];
	} data;

	__device__ void Initialise(xThreadInfo __xThreadInfo, xGenomeSet *__xGenomeSet);
	__device__ bool Assemble(xThreadInfo __xThreadInfo, xGenomeSet *__xGenomeSet);
        __device__ bool Assemble_PreProcess(xThreadInfo __xThreadInfo, xGenomeSet *__xGenomeSet);
	__device__ bool Assemble_PostProcess(xThreadInfo __xThreadInfo, xGenomeSet *__xGenomeSet);
	__device__ bool Assemble_Movelist(xThreadInfo __xThreadInfo, xGenomeSet *__xGenomeSet);
	__device__ bool Assemble_InPlace(xThreadInfo __xThreadInfo, xGenomeSet *__xGenomeSet);
	__device__ float fEvaluateFitness(xThreadInfo __xThreadInfo);//, bool __bSingleBlockId);
	//__device__ float fEvaluateFitnessForSingleGrid(xThreadInfo __xThreadInfo, xFitnessGrid *__xSingleFitnessGrid, bool __bIsSingleBlock);
	__device__ bool bSynchronizeBank(xThreadInfo __xThreadInfo);
};

__device__ float xAssembly::fEvaluateFitness(xThreadInfo __xThreadInfo){//, bool __bSingleBlockId){
        if(__xThreadInfo.WarpId()==0){
            this->data.gravity[__xThreadInfo.BankId()].x=0;
            this->data.gravity[__xThreadInfo.BankId()].y=0;
        }
        __syncthreads();
        //Step1: Evaluate Center of gravity
        short offset = (m_fit_DimGridX*m_fit_DimGridY) % mBankSize;
        short myshare = (m_fit_DimGridX*m_fit_DimGridY - offset) / mBankSize;
        short off_x=0, off_y=0;
        int sum_x=0, sum_y=0;
        for(int i=0;i<myshare;i++){
                off_x = (myshare*__xThreadInfo.WarpId()+i) % m_fit_DimGridX;
                off_y = (myshare*__xThreadInfo.WarpId()+i-off_x) / m_fit_DimGridX;
                if(this->data.grid.data.multi_d[off_x][off_y][this->data.flags[__xThreadInfo.BankId()].get_ucRed()][__xThreadInfo.BankId()].get_xCell()!=mEMPTY_CELL){
                     sum_x += off_x;
                     sum_y += off_y; 
                }
        }
        if(__xThreadInfo.WarpId()==mBankSize-1){
                for(int i=0;i<offset;i++){
                     if(this->data.grid.data.multi_d[off_x][off_y][this->data.flags[__xThreadInfo.BankId()].get_ucRed()][__xThreadInfo.BankId()].get_xCell()!=mEMPTY_CELL){
                           sum_x += off_x;
                           sum_y += off_y;
                     }
                }
        }
        __syncthreads();
        atomicAdd(&this->data.gravity[__xThreadInfo.BankId()].x, sum_x);
        atomicAdd(&this->data.gravity[__xThreadInfo.BankId()].y, sum_y);
        __syncthreads();
        //this->data.gravity[__xThreadInfo.BankId()].x=9;
        //this->data.gravity[__xThreadInfo.BankId()].y=8;
        /*if(__xThreadInfo.WarpId()==0){
                this->gravity_x[__xThreadInfo.BankId()] /= this->assembly_size[__xThreadInfo.BankId()];
                this->gravity_y[__xThreadInfo.BankId()] /= this->assembly_size[__xThreadInfo.BankId()];
        }
        __syncthreads();
        sum_x = 0;
        sum_y = 0;
        //Calculate (x,y) distances
        for(int i=0;i<myshare;i++){
                off_x = (myshare*__xThreadInfo.WarpId()+i) % mDimGridX;
                off_y = (myshare*__xThreadInfo.WarpId()+i-off_x) / mDimGridX;
                if(this->data.multi_d[off_x][off_y][__red][__xThreadInfo.BankId()].get_xCell()!=mEMPTY_CELL){
                     sum_x += (off_x - this->gravity_x[__xThreadInfo.BankId()])*(off_x - this->gravity_x[__xThreadInfo.BankId()]);
                     sum_y += (off_y - this->gravity_y[__xThreadInfo.BankId()])*(off_y - this->gravity_y[__xThreadInfo.BankId()]);
                }
        }
        if(__xThreadInfo.WarpId()==mBankSize-1){
                for(int i=0;i<offset;i++){
                     if(this->data.multi_d[off_x][off_y][__red][__xThreadInfo.BankId()].get_xCell()!=mEMPTY_CELL){
                     sum_x += (off_x - this->gravity_x[__xThreadInfo.BankId()])*(off_x - this->gravity_x[__xThreadInfo.BankId()]);
                     sum_y += (off_y - this->gravity_y[__xThreadInfo.BankId()])*(off_y - this->gravity_y[__xThreadInfo.BankId()]);
                     }
                }
        } 
        __syncthreads();
        atomicAdd(&this->shape_x[__xThreadInfo.BankId()], sum_x);
        atomicAdd(&this->shape_y[__xThreadInfo.BankId()], sum_y);
        __syncthreads();
        if(__xThreadInfo.WarpId()==0){
                this->shape_x[__xThreadInfo.BankId()] /= this->assembly_size[__xThreadInfo.BankId()];
                this->shape_y[__xThreadInfo.BankId()] /= this->assembly_size[__xThreadInfo.BankId()];
        }
        // Finished classification.
	*/
}

__device__ void xAssembly::Initialise(xThreadInfo __xThreadInfo, xGenomeSet *__xGenomeSet) {
	unsigned char TmpRed = this->data.flags[__xThreadInfo.BankId()].get_ucRed() % m_fit_NrRedundancyGridDepth;
	this->data.grid.Initialise(__xThreadInfo, TmpRed);
	this->data.movelist.Initialise(__xThreadInfo);
}

__device__ bool xAssembly::Assemble(xThreadInfo __xThreadInfo, xGenomeSet *__xGenomeSet) {
	bool TmpFlag = false;
	this->data.flags[__xThreadInfo.BankId()].ClearAll();
	TmpFlag = true; 
	if (TmpFlag) {
                //if(__xThreadInfo.WarpId() == 0){
              		this->data.edgesort.Initialise(__xThreadInfo, __xGenomeSet); //TEST
                //}
                //__syncthreads();
		//this->Assemble_PostProcess(__xThreadInfo, __xGenomeSet);
		if (TmpFlag) {
			//for (int i = 0; (i < m_fit_NrRedundancyAssemblies) && (!this->data.flags[__xThreadInfo.BankId()].get_bUNDCondition()); i++) {
                        for (int i = 0; (i < m_fit_NrRedundancyAssemblies); i++) {
				this->Initialise(__xThreadInfo, __xGenomeSet); //Empty out assembly grid at red
                                __syncthreads();
                                if(__xThreadInfo.WarpId() == 0){
             				bool TmpController = this->Assemble_Movelist(__xThreadInfo, __xGenomeSet); //TEST
                                }
                                //__syncthreads();
/*				if (!TmpController) TmpController = this->Assemble_InPlace(__xThreadInfo, __xGenomeSet);
				if (!TmpController) {
					// Both assembly processes did not finish! (should NEVER happen)
					return false; //Always false - indicate assembly did not finish properly (should not happen!)
				}
				this->data.flags[__xThreadInfo.BankId()].set_Red(i); //Choose next assembly step!
*/
			}
			return true; //Always true - i.e. indicate assembly did finish (can still be UND, though)
		} else {
			return false; //Indicates that processing before assembly returned either single block, or UND
		}

	} else {
		return false; //Indicates that processing before assembly returned either single block, or UND
	}

}

__device__ bool xAssembly::Assemble_PreProcess(xThreadInfo __xThreadInfo, xGenomeSet *__xGenomeSet) {
	unsigned char TmpSameCounter = 0;

	//NOTE: This should work, however, not clear how to communicate that single tile without initialisation of grid!
	//Check if starting tile is not empty
	for (int j = 0; j < mNrTileOrientations; j++) {
		if (__xGenomeSet->get_xEdgeType(__xThreadInfo, m_fit_TileIndexStartingTile,
				j) == 0)
			TmpSameCounter++;
	}
	if (TmpSameCounter == 4) {
		this->data.grid.get_xCell(__xThreadInfo, m_fit_DimGridX / 2,
				m_fit_DimGridY / 2, 0);
		return true; //Have finished assembly - UND is false, but so is PreProcess (trigger)
	}

	//Replace tile doublettes by empty tiles
	//Works for any number of mNrTileOrientations and mBitLengthEdgeType <= 4 Byte!
	//Note: This would be faster (but more inflexible) if tile-wise accesses!
	TmpSameCounter = 0;
	unsigned char DBGVAL1, DBGVAL2, DBGVAL3;
	for (int k = 0; k < mNrTileTypes - 1; k++) { //Go through all Tiles X (except for last one)
		for (int i = k + 1; i < mNrTileTypes; i++) { //Go through all Tiles X_r to the right
			for (int j = 0; j < mNrTileOrientations; j++) { //Go through all X edges rots
				TmpSameCounter = 0;
				for (int l = 0; l < mNrTileOrientations; l++) { //Cycle through all X edges
					DBGVAL1 = __xGenomeSet->get_xEdgeType(__xThreadInfo, k, l);
					DBGVAL2 = __xGenomeSet->get_xEdgeType(__xThreadInfo, i, (j
							+ l) % mNrTileOrientations);
					if (__xGenomeSet->get_xEdgeType(__xThreadInfo, k, l)
							== __xGenomeSet->get_xEdgeType(__xThreadInfo, i, (j
									+ l) % mNrTileOrientations)) {
						TmpSameCounter++;
					}
				}
				if (TmpSameCounter == mNrTileOrientations) {
					//Have detected a doublette - replace with empty tile!!
					for (int l = 0; l < mNrTileOrientations; l++) {
						//__xGenomeSet->set_EdgeType(__xThreadInfo, i, l, 0); //TEST
					}
				}
			}
		}
	}
	return true;
}

__device__ bool xAssembly::Assemble_PostProcess(xThreadInfo __xThreadInfo, xGenomeSet *__xGenomeSet) {
	//Optional: start at first tile and see if it can connect to any degenerate entries in EdgeSort directly
	//Note: If we can refrain from assembly, then save time for grid initialisation!
	unsigned char TmpBondingCounter = 0;
	unsigned char TmpEdgeTypeLength = 0;
	for (int j = 0; j < mNrTileOrientations; j++) {
		TmpEdgeTypeLength = this->data.edgesort.get_xLength(__xThreadInfo, j);
		if (TmpEdgeTypeLength > 1) {
			this->data.flags[__xThreadInfo.BankId()].set_TrivialUND(); //TEST
			return false;
		} else if (TmpEdgeTypeLength == 0) {
			TmpBondingCounter++;
		}
	}

	if (TmpBondingCounter == 4) {
		//(Single-tile assembly: PostProcess return value is false, but UND is also false (trigger) )
		this->data.grid.set_xCell(__xThreadInfo, m_fit_DimGridX / 2, m_fit_DimGridY / 2, 0, 0);
		return false;
	}
	//Note: (Optional) Could now check for periodicity (can return to tile X first tile starting at X at same orientation)
	//Note: (Optional) Could now check for 2x2 assembly, etc (quite rare though)
	//NOTE: TODO, have to check in EdgeSort whether Tile is symmetric, i.e. then remove bonding orientations
	return true;
}

__device__ bool xAssembly::Assemble_Movelist(xThreadInfo __xThreadInfo, xGenomeSet *__xGenomeSet) {
	//Place tiletype 0 on center of grid
	this->data.grid.set_xCell(__xThreadInfo, m_fit_DimGridX / 2, m_fit_DimGridY / 2, 0, 0);
	//Add first four moves to movelist (even iff they might be empty)
	uchar2 X; //X be current position in grid
	X.x = m_fit_DimGridX / 2;
	X.y = m_fit_DimGridY / 2;
        //return false; //TEST

	this->data.movelist.bPush(__xThreadInfo, this->data.grid.xGetNeighbourCellCoords(X.x, X.y, (unsigned char) 0));
	this->data.movelist.bPush(__xThreadInfo, this->data.grid.xGetNeighbourCellCoords(X.x, X.y, (unsigned char) 1));
	this->data.movelist.bPush(__xThreadInfo, this->data.grid.xGetNeighbourCellCoords(X.x, X.y, (unsigned char) 2));
	this->data.movelist.bPush(__xThreadInfo, this->data.grid.xGetNeighbourCellCoords(X.x, X.y, (unsigned char) 3));

        //return false; //TEST
        
	//We use movelist approach to assemble grids
	//Will switch to in-place assembly if either movelist full, or some other pre-defined condition.

	//Note: If we want mixed redundancy detection, need to implement some Single-Assembly Flag in AssemblyFlags that will switch.
	//Also: SynchronizeBank() needs to be adapted to not wait for other threads iff Many-thread approach!

#ifndef m_fit_MULTIPLE_WARPS
	xCell N; //N(E_X) be non-empty neighbouring cells
	unsigned char Mirr; // Mirr(E_X, N(E_X)) be tile edge neighbouring E_X
	xCell T, TmpT; // T(Mirr(E_X, N(E_X)) be potential bonding tiles
	//For all elements M in Movelist (and while not UND condition detected)
	while ((this->data.movelist.get_sPos(__xThreadInfo) >= 0) && (!this->data.flags[__xThreadInfo.BankId()].get_bUNDCondition())) {
		//Choose position X from Movelist and remove it from Movelist

		//this->data.grid.print(__xThreadInfo, __xGenomeSet);
                //return false;
		X = this->data.movelist.xPop(__xThreadInfo);
                //return false;
		T.set_xCell(mEMPTY_CELL);
		for (int E_X = 0; (E_X < mNrTileOrientations)
				&& (!this->data.flags[__xThreadInfo.BankId()].get_bUNDCondition()); E_X++) {
			//::Let N(E_X) be non-empty neighbouring cells.
			N = this->data.grid.xGetNeighbourCell(__xThreadInfo, X.x, X.y, this->data.flags[__xThreadInfo.BankId()].get_ucRed(), (unsigned char) E_X);
			if (N.get_xCell() != mEMPTY_CELL) { //For all N(E_X)
				//::Let Mirr(E_X, N(E_X)) be tile neighbouring E_X
				//Mirr = __xGenomeSet->get_xEdgeType(__xThreadInfo, N.get_xType(), (mNrTileOrientations-(E_X+mNrTileOrientations/2)%mNrTileOrientations)%mNrTileOrientations );
				unsigned char DBGVAL = N.get_xOrient();
				unsigned char TmpMirrorCoord = (4 - N.get_xOrient() + (E_X + mNrTileOrientations / 2) % mNrTileOrientations) % mNrTileOrientations;
				Mirr = __xGenomeSet->get_xEdgeType(__xThreadInfo, N.get_xType(), TmpMirrorCoord);
				//For all Mirr(E_X, N(E_X)), let T(Mirr(E_X, N(E_X)) be potential bonding tiles
				TmpT.set_xCell(this->data.edgesort.GetBondingTile( __xThreadInfo, Mirr, &this->data.states[__xThreadInfo.BankId()], &this->data.flags[__xThreadInfo.BankId()]));

				//NOTE: TrivialUND can arise in three ways:
				//1. For some Mirr, there is more than 1 bonding tile T (TrivialUND raised by GetBondingTile)
				//2. For some T, there is more than one orientation O
				//3. T does not agree between all N
				//Else if | T( Mirr( E_X, N(E_X) ) ) | == 0
				//If | T( Mirr( E_X, N(E_X) ) ) | > 0
				//Raise TrivialUND condition
				//Else If | T( Mirr( E_X, N(E_X) ) ) | == 1
				//if ( T.get_xCell() != mEMPTY_CELL ){ //Check if already tile there ??
				if (TmpT.get_xCell() != mEMPTY_CELL) {
					//if( TmpT.get_xCell() != T.get_xCell() ){
					//	//Raise TrivialUND!
					//	this->data.flags[__xThreadInfo->WarpId()].set_TrivialUND();
					//}
					T.set_xCell(TmpT.get_xCell());
					//As Bonding Cell is rotated such that bonding edge is facing North,
					//we need to rotate tile T such that bonding edge faces bonding site
					//Note: bonding orientations are handled above (GetBondingTile includes orientation).
					//::Let O(T) be all bonding orientations of T
					//If |O(T)| > 1
					//Else If |O(T)| = 1 --> Check Steric, if not --> Assemble
					//Let T* be T rotated such that E_T*(E_X) == E_T(O(T))
					//unsigned char DBGVAL10 = (T.get_xOrient() + E_X) % mNrTileOrientations;
					//unsigned char DBGVAL11 = T.get_xOrient();
					//printf("CELL DBG %d and Orient: %d:\n", T.get_xCell(), T.get_xOrient());
					T.set_Orient((T.get_xOrient() + E_X) % mNrTileOrientations);
				}
			}
		}
		if (!this->data.flags[__xThreadInfo.BankId()].get_bUNDCondition() && T.get_xCell() != mEMPTY_CELL) {

			//NOTE: StericUND can arise in two ways:
			//1. T does not agree with tile from previous assembly run
			//2. T does not agree with tile already at X in same run (multiple threads only)
#ifdef m_fit_MULTIPLE_WARPS
			//NOTE: Multi-threading only: Check if there is already a different non-empty tile at X!
			TmpT = this->data.grid.get_xCell(__xThreadInfo, X.x, X.y, this->flags.get_ucRed());
			if(TmpT.get_xCell() != mEMPTY_CELL) {
				if(TmpT.get_xCell() != T.get_xCell()) {
					this->data.flags[__xThreadInfo.BankId()].set_StericUND(); //TEST
				}
			}
#endif

			if (this->data.flags[__xThreadInfo.BankId()].get_ucRed()) {
				TmpT = this->data.grid.get_xCell(__xThreadInfo, X.x, X.y, this->data.flags[__xThreadInfo.BankId()].get_ucRed() - 1);
				if (TmpT.get_xCell() != T.get_xCell()) { //We have detected steric non-determinism!
					this->data.flags[__xThreadInfo.BankId()].set_StericUND(); //TEST
				}
			}
			if (!this->data.flags[__xThreadInfo.BankId()].get_bUNDCondition()) {
				//If X is not BorderCell
				//Assemble T* at X
				//Note: set_xCell will return false if BorderCell case!
				if (T.get_xCell() != mEMPTY_CELL) {
					if (!this->data.grid.set_xCell(	__xThreadInfo, X.x, X.y, this->data.flags[__xThreadInfo.BankId()].get_ucRed(), T.get_xCell())) {
						this->data.flags[__xThreadInfo.BankId()].set_UnboundUND();
					}
					if (!this->data.flags[__xThreadInfo.BankId()].get_bUNDCondition()) {
						xFourPermutation TmpAddPerm((int) (hiprand_uniform(
								&this->data.states[__xThreadInfo.BankId()])
								* 24.0f));
						unsigned char E_X;
						while (TmpAddPerm.bNotTraversed()) {
							E_X = TmpAddPerm.ucWalk();
							unsigned char DBGVAL = TmpAddPerm.WalkIndex;
							//For all n(E_X)
							N = this->data.grid.xGetNeighbourCell(__xThreadInfo, X.x, X.y, this->data.flags[__xThreadInfo.BankId()].get_ucRed(), (unsigned char) E_X);
							//::Let n(E_X) be empty neighbour cells.
							if (N.get_xCell() == mEMPTY_CELL) {
								if (!this->data.movelist.bPush(	__xThreadInfo, this->data.grid.xGetNeighbourCellCoords(X.x, X.y, (unsigned char) E_X))) {
									this->data.flags[__xThreadInfo.BankId()].set_BusyFlag();
								}
							}
						}
					}
				}
			}
		}
	}
	if (!this->data.flags[__xThreadInfo.BankId()].get_bBusyFlag())
		return true;
	else
		return false; //i.e. Need to continue with in-place assembly!
#else
#endif
}

__device__ unsigned char xEdgeSort::GetBondingTile(xThreadInfo __xThreadInfo,
                short __sEdgeId, hiprandState *__xCurandState,
                xAssemblyFlags *__xAssemblyFlags) {
        //Takes: Edge Type to which the tile should bond, FitFlags which will be set according to UND conditions
        //Returns: Cell of Bonding Tile type which is rotated such that the bonding tile is facing NORTH (0),
        //If nothing bonds, will return mEMPTY_CELL instead.
        if (this->get_xLength(__xThreadInfo, __sEdgeId) == 1) {
                xCell TmpCell;
                unsigned char DBGVAL2, DBGVAL3, DBGVAL = GetBondingTileOrientation(
                                __xThreadInfo, __sEdgeId, 0, __xAssemblyFlags);
                unsigned char TmpBondBuffer = GetBondingTileOrientation(__xThreadInfo,
                                __sEdgeId, 0, __xAssemblyFlags);
                TmpCell.set_xCell(4 - TmpBondBuffer);
                TmpCell.set_Type(this->get_xData(__xThreadInfo, __sEdgeId, 0,
                                TmpBondBuffer)); //TEST (0 anstelle TmpCell.get_xOrient()) b-fore
                return TmpCell.get_xCell();
        } else if (this->get_xLength(__xThreadInfo, __sEdgeId) == 0) {
                return mEMPTY_CELL;
        } else {
                __xAssemblyFlags->set_TrivialUND();
                return mEMPTY_CELL;
        }
}

__device__ unsigned char xEdgeSort::GetBondingTileOrientation(xThreadInfo __xThreadInfo, unsigned char __ucEdgeId, unsigned char __ucTileId, xAssemblyFlags *__xAssemblyFlags) {
	unsigned char TmpCounter = 0, TmpTile, TmpOrient = mEMPTY_CELL;
	for (int i = 0; i < mNrTileOrientations; i++) {
		TmpTile = this->get_xData(__xThreadInfo, __ucEdgeId, __ucTileId, i);
		if (TmpTile != mEMPTY_CELL) {
			TmpOrient = i;
			TmpCounter++;
			if (TmpCounter >= 2) {
				__xAssemblyFlags->set_TrivialUND();
				break;
			}
		}
	}
	return TmpOrient; //should never be mEMPTY_CELL!
	//Returns edge-id of neighbouring tile that bonds
}

__device__ unsigned char xEdgeSort::get_xData(xThreadInfo __xThreadInfo, unsigned char __ucEdgeId, unsigned char __ucTileId, unsigned char __ucOrientation) {
	return this->data.multi_d[__ucEdgeId][__ucTileId][__ucOrientation][__xThreadInfo.BankId()];
}

__device__ bool xAssembly::Assemble_InPlace(xThreadInfo __xThreadInfo,	xGenomeSet *__xGenomeSet) {
        return true;
}

}

__global__ void TestAssemblyKernel(unsigned char *g_ucGenomes, float *g_ucFitnessValues, unsigned char *g_ucGrids, hiprandState *states)
{
    __shared__ xGenomeSet s_xGenomeSet;
    //__shared__ xEdgeSort s_xEdgeSort;
    __shared__ xAssembly s_xAssembly;
    s_xAssembly.data.states = states;
    xThreadInfo r_xThreadInfo(threadIdx.x, threadIdx.y, blockIdx.x, blockIdx.y);
    s_xGenomeSet.CopyFromGlobal(r_xThreadInfo, g_ucGenomes);
    //s_xEdgeSort.Initialise(r_xThreadInfo, &s_xGenomeSet, -1);
    s_xAssembly.Assemble(r_xThreadInfo, &s_xGenomeSet);
    for(int i=0;i<4;i++){
        //s_xGenomeSet.data.multi_d[r_xThreadInfo.BankId()].data.one_d[i] = s_xEdgeSort.length.multi_d[i][r_xThreadInfo.BankId()];
        //s_xGenomeSet.data.multi_d[r_xThreadInfo.BankId()].data.one_d[i] = s_xEdgeSort.data.multi_d[6][0][i][r_xThreadInfo.BankId()];
        //s_xGenomeSet.data.multi_d[r_xThreadInfo.BankId()].data.one_d[i] = tex2D(t_ucInteractionMatrix, i, 1);
    }
    s_xGenomeSet.CopyToGlobal(r_xThreadInfo, g_ucGenomes); 
    for(int i=0;i<m_fit_DimGridY;i++){
	 for(int j=0;j<m_fit_DimGridX;j++){
             xCell TMP = s_xAssembly.data.grid.get_xCell(r_xThreadInfo, i, j, 0);
             g_ucGrids[r_xThreadInfo.BankId()*m_fit_DimGridX*m_fit_DimGridY + j*m_fit_DimGridX + i] = s_xAssembly.data.grid.get_xCell(r_xThreadInfo, i, j, 0).get_xType();
         }
    }
}


__global__ void SearchSpaceKernel(unsigned char *g_ucGenomes,  unsigned char *g_ucGrids, int *g_ucFitnessLeft, int *g_ucFitnessBottom,  hiprandState *states)
{
    __shared__ xGenomeSet s_xGenomeSet;
    //__shared__ xEdgeSort s_xEdgeSort;
    __shared__ xAssembly s_xAssembly;
    s_xAssembly.data.states = states;
    xThreadInfo r_xThreadInfo(threadIdx.x, threadIdx.y, blockIdx.x, blockIdx.y);
    //s_xGenomeSet.CopyFromGlobal(r_xThreadInfo, g_ucGenomes);
    //s_xEdgeSort.Initialise(r_xThreadInfo, &s_xGenomeSet, -1);

    s_xGenomeSet.data.multi_d[r_xThreadInfo.BankId()].data.one_d[0] = 40;
    s_xGenomeSet.data.multi_d[r_xThreadInfo.BankId()].data.one_d[1] = 0;
    s_xGenomeSet.data.multi_d[r_xThreadInfo.BankId()].data.one_d[2] = 0;
    s_xGenomeSet.data.multi_d[r_xThreadInfo.BankId()].data.one_d[3] = 0;

    s_xAssembly.Assemble(r_xThreadInfo, &s_xGenomeSet);
    s_xAssembly.fEvaluateFitness(r_xThreadInfo);

    if(r_xThreadInfo.WarpId()==0){
        //g_ucFitnessLeft[r_xThreadInfo.GlobId(1)] = 3; //s_xAssembly.data.gravity[r_xThreadInfo.BankId()].x;
        g_ucFitnessLeft[(blockIdx.y*m_fit_DimBlockX + blockIdx.x)*32+r_xThreadInfo.BankId()] = s_xAssembly.data.gravity[r_xThreadInfo.BankId()].x;
// (blockIdx.y*m_fit_DimBlockX + blockIdx.x)*32+r_xThreadInfo.BankId();//s_xAssembly.data.gravity[r_xThreadInfo.BankId()].x;
        g_ucFitnessBottom[(blockIdx.y*m_fit_DimBlockX + blockIdx.x)*32+r_xThreadInfo.BankId()] = s_xAssembly.data.gravity[r_xThreadInfo.BankId()].y;
    //g_ucFitnessBottom[462] = 7;
    }

    //for(int i=0;i<4;i++){
        //s_xGenomeSet.data.multi_d[r_xThreadInfo.BankId()].data.one_d[i] = s_xEdgeSort.length.multi_d[i][r_xThreadInfo.BankId()];
        //s_xGenomeSet.data.multi_d[r_xThreadInfo.BankId()].data.one_d[i] = s_xEdgeSort.data.multi_d[6][0][i][r_xThreadInfo.BankId()];
        //s_xGenomeSet.data.multi_d[r_xThreadInfo.BankId()].data.one_d[i] = tex2D(t_ucInteractionMatrix, i, 1);
    //}
    //s_xGenomeSet.CopyToGlobal(r_xThreadInfo, g_ucGenomes);

    //Copy to grid
    for(int i=0;i<m_fit_DimGridY;i++){
         for(int j=0;j<m_fit_DimGridX;j++){
             xCell TMP = s_xAssembly.data.grid.get_xCell(r_xThreadInfo, i, j, 0);
             g_ucGrids[r_xThreadInfo.FlatBlockId()*m_fit_DimGridX*m_fit_DimGridY*32 + r_xThreadInfo.BankId()*m_fit_DimGridX*m_fit_DimGridY + j*m_fit_DimGridX + i] = s_xAssembly.data.grid.get_xCell(r_xThreadInfo, i, j, 0).get_xType();
         }
    }
}

