#include "hip/hip_runtime.h"
//#include <stdio.h>

{# {% include "header_inc.cuh" %} #}
{# {% include "fit_header_inc.cuh" %} #}

struct xThreadInfo {
        ushort4 data;
        
        __device__  xThreadInfo(unsigned short __usThreadIdX, unsigned short __usThreadIdY, unsigned short __usBlockIdX, unsigned short __usBlockIdY);
        __device__ unsigned short WarpId(void);
        __device__ unsigned short BankId(void);
        __device__ unsigned short FlatThreadId(void);
        __device__ unsigned short FlatBlockId(void);
        __device__ unsigned short GlobId(unsigned short __usTypeLength);
        __device__ void __DEBUG_CALL(void);
};

__device__ xThreadInfo::xThreadInfo(unsigned short __usThreadIdX, unsigned short __usThreadIdY, unsigned short __usBlockIdX, unsigned short __usBlockIdY) {
        this->data.z = threadIdx.y * m_fit_DimThreadX + threadIdx.x; //Flat Thread ID
        this->data.x = this->data.z % mWarpSize; //BankID
        this->data.y = (this->data.z - this->data.x) / mWarpSize; //WarpID
        this->data.w = blockIdx.y * m_fit_DimBlockX + blockIdx.x; //Flat Block ID
}

__device__ unsigned short xThreadInfo::WarpId(void) {
        return this->data.y;
}
__device__ unsigned short xThreadInfo::BankId(void) {
        return this->data.x;
}
__device__ unsigned short xThreadInfo::FlatThreadId(void) {
        return this->data.z;
}
__device__ unsigned short xThreadInfo::FlatBlockId(void) {
        return this->data.w;
}

__device__ unsigned short xThreadInfo::GlobId(unsigned short __usTypeLength) {
        return (this->data.w * m_fit_DimThreadX * m_fit_DimThreadY + this->data.z) * __usTypeLength;
}

struct xGenome {
        union{
            unsigned char one_d[mAlignedByteLengthGenome];
        } data;

        __device__ void CopyFromGlobal(xThreadInfo __xThreadInfo, unsigned char *__g_ucGenomeSet);
	__device__ void CopyToGlobal(xThreadInfo __xThreadInfo, unsigned char *__g_ucGenomeSet);
        __device__ unsigned char get_xEdgeType(unsigned char __ucTileId, unsigned char __ucEdgeId);
        __device__ void set_EdgeType(xThreadInfo *__xThreadInfo, unsigned char __ucTileId, unsigned char __ucEdgeId, unsigned char __ucVal);
};

struct xGenomeSet {
        union{        
            xGenome multi_d[mWarpSize];        
            unsigned char one_d[mWarpSize*sizeof(xGenome)];
        } data;

        __device__ void CopyFromGlobal(xThreadInfo __xThreadInfo, unsigned char *__g_ucGenomeSet);
        __device__ void CopyToGlobal(xThreadInfo __xThreadInfo, unsigned char *__g_ucGenomeSet);
        __device__ unsigned char get_xEdgeType(xThreadInfo __xThreadInfo, unsigned char __ucTileId, unsigned char __ucEdgeId);
        __device__ unsigned char set_EdgeType(xThreadInfo *__xThreadInfo, unsigned char __ucTileId, unsigned char __ucEdgeId, unsigned char __ucVal);
        __device__ void print(xThreadInfo *__xThreadInfo);
};

__device__ void xGenomeSet::CopyFromGlobal(xThreadInfo __xThreadInfo, unsigned char *__g_ucGenomes) {
        this->data.multi_d[__xThreadInfo.BankId()].CopyFromGlobal(__xThreadInfo, __g_ucGenomes);
}

__device__ void xGenome::CopyFromGlobal(xThreadInfo __xThreadInfo, unsigned char *__g_ucGenomes) {
        for (int i = 0; i < mAlignedByteLengthGenome; i ++) {
             this->data.one_d[i] = __g_ucGenomes[__xThreadInfo.GlobId(sizeof(xGenome)) + i];
        }
}

__device__ void xGenome::CopyToGlobal(xThreadInfo __xThreadInfo, unsigned char *__g_ucGenomes) {
        for (int i = 0; i < mAlignedByteLengthGenome; i += 1) {
                //(*reinterpret_cast<int*> (&this->data.one_d[i])) = (*reinterpret_cast<int*> (&__g_ucGenomeSet[__xThreadInfo->GlobId(sizeof(xGenome)) + i]));
                __g_ucGenomes[__xThreadInfo.GlobId(sizeof(xGenome)) + i] = this->data.one_d[i];
        }
}

__device__ void xGenomeSet::CopyToGlobal(xThreadInfo __xThreadInfo, unsigned char *__g_ucGenomes) {
        this->data.multi_d[__xThreadInfo.BankId()].CopyToGlobal(__xThreadInfo, __g_ucGenomes);
}

__device__ unsigned char xGenome::get_xEdgeType( unsigned char __ucTileId, unsigned char __ucEdgeId){
    if (__ucTileId < mNrTileTypes) {
        unsigned short TmpStartBit = __ucTileId * mBitLengthEdgeType * mNrTileOrientations + __ucEdgeId * mBitLengthEdgeType;
	unsigned short TmpEndBit = TmpStartBit + mBitLengthEdgeType;
	unsigned char TmpRetVal = 0;
	unsigned short TmpByteOffset = 0;
	unsigned short TmpBitOffset = 0;
	//Note: This could be speeded up by copying all bits within a byte simultaneously
	unsigned short j = 0;
	for (int i = TmpStartBit; i < TmpEndBit; i++) {
	    TmpBitOffset = i % 8; //We need to invert index as we start from left to right
	    TmpByteOffset = (i - TmpBitOffset) / 8;
	    TmpBitOffset = 7 - TmpBitOffset;
	    TmpRetVal += mOneOrNil(mBitTest(TmpBitOffset, this->data.one_d[TmpByteOffset])) << (mBitLengthEdgeType - 1 - j);
	    j++;
            //return mBitTest(TmpBitOffset,8);
	}
        return TmpRetVal;
    } else return (unsigned char) 0x00;
}

__device__ unsigned char xGenomeSet::get_xEdgeType(xThreadInfo __xThreadInfo, unsigned char __ucTileId, unsigned char __ucEdgeId){
    return this->data.multi_d[__xThreadInfo.BankId()].get_xEdgeType(__ucTileId, __ucEdgeId);
}

//GENOME TEST KERNEL
__global__ void TestGenomeKernel(unsigned char *dest)
{
    __shared__ xGenomeSet Tmp;
    xThreadInfo Tmpa(threadIdx.x, threadIdx.y, blockIdx.x, blockIdx.y);
    Tmp.CopyFromGlobal(Tmpa, dest);   
    for(int i=0;i<4;i++){
        Tmp.data.multi_d[Tmpa.BankId()].data.one_d[i] = Tmp.get_xEdgeType(Tmpa, i, 1);
    }
    Tmp.CopyToGlobal(Tmpa, dest);
}
