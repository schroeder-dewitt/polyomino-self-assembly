#include "hip/hip_runtime.h"
extern "C" __global__ void CurandInitKernel(hiprandState *state)
{
    int id = m_curand_NR_THREADS_PER_BLOCK * mBLOCK_ID + mTHREAD_ID;
    //Each thread gets same seed, a different sequence number, no offset 
    hiprand_init(1234, id, 0, &state[id]);
}
