#include "hip/hip_runtime.h"
/* -----------------------------------------------------------------------------
                Declare Textures and Constant Memory
----------------------------------------------------------------------------- */


//texture<float, 2> t_ucInteractionMatrix;

__constant__ float c_fParams[28];
__constant__ float c_fFitnessParams[48];
__constant__ float c_ucFourPermutations[24][4];
__constant__ float c_fFitnessSumConst;
__constant__ float c_fFitnessListConst[512];
__constant__ float c_fGAParams[52];


/* -----------------------------------------------------------------------------
                Include All Header Files
----------------------------------------------------------------------------- */
#include "hiprand/hiprand_kernel.h"

extern "C"
{

//start include globals.inc.cuh
typedef int xMutex;
typedef unsigned char ucTYPELENGTH; 
typedef unsigned short usSHARED1D;
typedef unsigned int usGLOBAL1D;

enum Params {
    eNrGenomes,
    eNrGenerations,
    eNrTileTypes,
    eNrEdgeTypes,
    eByteLengthGenome,
    eBitLengthGenome,
    eEdgeTypeBitLength,
    eNrTileOrientations
};

#define mXOR(a, b) (((a)&~(b))|(~(a)&(b)))

#define mBLOCK_ID blockIdx.x
#define mTHREAD_ID_X threadIdx.x
#define mTHREAD_ID_Y threadIdx.y
#define mTHREAD_ID threadIdx.x
#define mNrMemoryBanks warpSize

#define mByteLengthGenome_c c_fParams[eByteLengthGenome]
#define mNrGenomes_c c_fParams[eNrGenomes]
#define mNrGenomes 512
#define mBitLengthGenome_c c_fParams[eBitLengthGenome]
#define mBitLengthEdgeType_c c_fParams[eEdgeTypeBitLength]
#define mNrTileTypes_c c_fParams[eNrTileTypes]
#define mNrEdgeTypes_c c_fParams[eNrEdgeTypes]
#define mNrTileOrientations_c c_fParams[eNrTileOrientations]
                
#define mNrTileTypes 4
#define mNrEdgeTypes 8
#define mNrTileOrientations 4
#define mByteLengthGenome 4
#define mBitLengthGenome 32
#define mBitLengthEdgeType 3

/*This file contains all template macros for global simulation - Copyright Christian Schroeder, Oxford University 2012*/

//#define SAFE_MEMORY_MAPPING 
#define mAlignedByteLengthGenome 4
#define mWarpSize 32
#define mBankSize 8
//#define mNrTileOrientations 4
//#define mBitLengthEdgeType 
//#define mNrTileTypes 
//#define mNrEdgeTypes 

//#define EMPTY_TILETYPE 63
//end include globals.inc.cuh

//start include curandinit.inc.cuh
#define m_curand_NR_THREADS_PER_BLOCK 256.0

/*extern "C" __global__ void CurandInitKernel(hiprandState *state);*/
//end include curandinit.inc.cuh

/*This header file includes all template macros for the Fitness Kernel - copyright Christian Schroeder, Oxford University, 2012*/

#define m_fit_DimThreadX 
#define m_fit_DimThreadY 
#define m_fit_DimBlockX 

#define m_fit_DimGridX 
#define m_fit_DimGridY 

#define mFFOrNil(param) (param?0xFF:0x00)
#define mOneOrNil(param) (param?0x01:0x00)
#define mBitTest(index,byte) (byte & (0x1<<index))

#define m_fit_LengthMovelist 
#define m_fit_NrRedundancyGridDepth 
#define m_fit_NrRedundancyAssemblies 
#define m_fit_TileIndexStartingTile 

#define mEMPTY_CELL 255
#define mEMPTY_CELL_ML 22<<2 //254
#define mEMPTY_CELL_OUT_OF_BOUNDS 253

//start include sorting.inc.cuh
#define m_sorting_NR_THREADS_PER_BLOCK 256

/*__global__ void SortingKernel(float *g_fFFValues);*/
//end include sorting.inc.cuh

//start include ga_utils.inc.cuh

/*TESTED*/
__forceinline__ __device__ unsigned int ga_uiPoissonDistribution(float r_fMean, hiprandState *g_xCurandState){
    #ifdef EFFICIENT_POISSON
    /* Implement some fast algorithm (see logbook for one based on rejection) and / or use cumulative probability table
       loaded from constant memory
    */
    #else
    //Knuth implementation
    float L = expf(-r_fMean); 
    float p = 1.0f; 
    int k = 0; 
    
    do { 
        k++; 
        p *= hiprand_uniform(g_xCurandState); 
    } while (p > L); 
    return k - 1;
    #endif
} 
//end include ga_utils.inc.cuh

//start include ga.inc.cuh
#define m_ga_ProbabilityUniformCrossover_c c_fGAParams[e_ga_UniformCrossoverProbability]
#define m_ga_ProbabilitySinglePointCrossover_c c_fGAParams[e_ga_SinglePointCrossoverProbability]
#define m_ga_RateMutation_c c_fGAParams[e_ga_RateMutation]
#define m_ga_NR_THREADS_PER_BLOCK 256
#define m_ga_THREAD_DIM_X 256
#define m_ga_THREAD_DIM_Y 1

enum GAParams {
    e_ga_RateMutation,
    e_ga_ProbabilityUniformCrossover,
    e_ga_ProbabilitySinglePointCrossover,
    e_ga_FlagMixedCrossover
};


//extern "C" __global__ void GAKernel(unsigned char *g_ucGenomes, float *g_fFFValues, unsigned char *g_ucAssembledGrids, hiprandState *g_xCurandStates);



#define WITH_BANK_CONFLICT


#define WITH_NAIVE_ROULETTE_WHEEL_SELECTION



#define WITH_ASSUME_NORMALIZED_FITNESS_FUNCTION_VALUES



#define WITH_SINGLE_POINT_CROSSOVER


#define WITH_SUREFIRE_MUTATION


//TESTED
__forceinline__ __device__ unsigned int ga_uiSelectionRouletteWheel(hiprandState *state){
    unsigned int r_uiCutoffIndex = 0;
    float r_fPartSum = 0.0f;
    float r_fRandomNumber = hiprand_uniform(state);
    for(int i=0;i<mNrGenomes;i++){ //Perform Roulette Wheel Selection on Constant Memory
        r_fPartSum += c_fFitnessListConst[i] / c_fFitnessSumConst;
        if(r_fPartSum >= r_fRandomNumber){
            r_uiCutoffIndex = i;
            break;
        }
    }    
    return r_uiCutoffIndex;
}

//TESTED
__forceinline__ __device__ void ga_CrossoverUniform( unsigned char (&s_ucGenome)[m_ga_NR_THREADS_PER_BLOCK][mByteLengthGenome],
                                                     unsigned char *g_ucGenomes,
                                                     unsigned int r_uiCutoffIndex,
                                                     hiprandState *state ){
    unsigned int r_uiNumberOfRandCallsRequired = (unsigned int) ( (mByteLengthGenome - mByteLengthGenome % sizeof(float))/sizeof(float) + 1 ); //Establish number of rand calls required
    float r_fRandBuffer;
    unsigned char r_ucRandMask;
    unsigned int r_uiIndexBuffer;
    for(int i=0;i<r_uiNumberOfRandCallsRequired;i++){
        r_fRandBuffer = hiprand_uniform(state); 
        
        for(int j=0;j<sizeof(float);j++){      
            r_ucRandMask = (unsigned int) ((reinterpret_cast<int&>(r_fRandBuffer) & (0xFF << 8 * j)) >> 8 * j); //Select next byte from r_fRandBuffer
            r_uiIndexBuffer = i*sizeof(float) + j;
            if(r_uiIndexBuffer < mByteLengthGenome){ 
                /*NOTE: If we decide to pad global memory or so, then we have to adjust these functions here!*/
                s_ucGenome[mTHREAD_ID][r_uiIndexBuffer] = (s_ucGenome[mTHREAD_ID][r_uiIndexBuffer] & r_ucRandMask) + (g_ucGenomes[r_uiCutoffIndex * mByteLengthGenome + r_uiIndexBuffer] & (~r_ucRandMask)) ;
            }    
        }          
        //printf("\n");
    }
}

//TESTED
__forceinline__ __device__ void ga_CrossoverSinglePoint( unsigned char (&s_ucGenome)[m_ga_NR_THREADS_PER_BLOCK][mByteLengthGenome],
                                                         unsigned char *g_ucGenomes,
                                                         unsigned int r_uiCutoffIndex,
                                                         hiprandState *state){
    unsigned short int r_uiCrossoverPoint = hiprand_uniform(state) * mBitLengthGenome;
    unsigned short int r_uiCrossoverBitOffset = r_uiCrossoverPoint % 8;
    unsigned short int r_uiCrossoverByte = (r_uiCrossoverPoint - r_uiCrossoverBitOffset) / 8;
    unsigned int r_uiIndexBuffer;
    for(int j=0;j<=r_uiCrossoverByte;j++){ 
        if(j == r_uiCrossoverByte){     
            s_ucGenome[mTHREAD_ID][j] = ( s_ucGenome[mTHREAD_ID][j] & ( 0xFF >> r_uiCrossoverBitOffset ) ) + ( g_ucGenomes[r_uiCutoffIndex * mByteLengthGenome + j] & ( 0xFF << ( 8 - r_uiCrossoverBitOffset )  ) );         
        } else {
            s_ucGenome[mTHREAD_ID][j] = g_ucGenomes[r_uiCutoffIndex * mByteLengthGenome + j];
        }
    }          
    //printf("SinglePoint: %d (Byte: %d, Offset: %d)\n", r_uiCrossoverPoint, (r_uiCrossoverPoint - r_uiCrossoverBitOffset)/8 , r_uiCrossoverBitOffset);  
}


//TESTED
__forceinline__ __device__ void ga_MutationSurefire( unsigned char (&s_ucGenome)[m_ga_NR_THREADS_PER_BLOCK][mByteLengthGenome],
                                                      float mutation_rate, 
                                                      hiprandState *state){
    float r_fRandBuf;
    for(int i=0;i<mByteLengthGenome;i++){
        for(int j=0;j<8;j++){
            r_fRandBuf = hiprand_uniform(state) * mBitLengthGenome; 
            if(r_fRandBuf <= mutation_rate){
                s_ucGenome[mTHREAD_ID][i] = mXOR(s_ucGenome[mTHREAD_ID][i], 1 << j);
            }
        }
    }
}


//TESTED
__forceinline__ __device__ void ga_MutationSophisticated( unsigned char (&s_ucGenome)[m_ga_NR_THREADS_PER_BLOCK][mByteLengthGenome],
                                                     unsigned char (&s_ucBufGenome)[m_ga_NR_THREADS_PER_BLOCK][mByteLengthGenome],
                                                     float mutation_rate, 
                                                     hiprandState *state){
    /* Note: this can be done more efficiently by just using 
       a dynamic list of mutated indices, however this only works for  > 2.0
       Of course, could try to work around this - however, actually probably performant enough!
    */ 
    for(int i=0;i<mByteLengthGenome;i++){
        s_ucBufGenome[mTHREAD_ID][i] = 0;
    }
    float r_fNrMutations = ga_uiPoissonDistribution(mutation_rate, state);
    if(r_fNrMutations > mBitLengthGenome) r_fNrMutations = mBitLengthGenome;
    float r_fRandBuf;
    short r_ssBitOffset, r_ssByteOffset;
    bool r_bRetry = false;
    for(int i=0;i<r_fNrMutations;i++){
        r_fRandBuf = hiprand_uniform(state) * mBitLengthGenome;
        r_ssBitOffset =  (signed short) r_fRandBuf % 8 ;
        r_ssByteOffset = ((signed short) r_fRandBuf - r_ssBitOffset) / 8;
        if( ! ( s_ucBufGenome[mTHREAD_ID][r_ssByteOffset] & (1 << r_ssBitOffset) ) ){ //bit set already?
            s_ucBufGenome[mTHREAD_ID][r_ssByteOffset] += (1 << r_ssBitOffset); //Set bit!
            r_bRetry=false;
        } else {
            i--;//Try again to mutate!
        }
    }
    for(int i=0;i<mByteLengthGenome;i++){
        s_ucGenome[mTHREAD_ID][i] = mXOR(s_ucGenome[mTHREAD_ID][i], s_ucBufGenome[mTHREAD_ID][i]);
    }
}
//end include ga.inc.cuh

/* -----------------------------------------------------------------------------
                Define Kernels
----------------------------------------------------------------------------- */
    
    //start include globals.inc.cu
__forceinline__ __device__ void u_Lock(int &mutex){
		while ( atomicCAS( &mutex, 0, 1) != 0);		
}

__forceinline__ __device__ void u_Unlock(int &mutex){
	atomicExch(&mutex, 0);
}
    //end include globals.inc.cu

    //start include curandinit.inc.cu
 extern "C" __global__ void CurandInitKernel(hiprandState *state)
{
    int id = m_curand_NR_THREADS_PER_BLOCK * mBLOCK_ID + mTHREAD_ID;
    //Each thread gets same seed, a different sequence number, no offset 
    hiprand_init(1234, id, 0, &state[id]);
} 
    //end include curandinit.inc.cu

    //start include fitness.cu
/*This header file includes all template macros for the Fitness Kernel - copyright Christian Schroeder, Oxford University, 2012*/

#define m_fit_DimThreadX 
#define m_fit_DimThreadY 
#define m_fit_DimBlockX 

#define m_fit_DimGridX 
#define m_fit_DimGridY 

#define mFFOrNil(param) (param?0xFF:0x00)
#define mOneOrNil(param) (param?0x01:0x00)
#define mBitTest(index,byte) (byte & (0x1<<index))

#define m_fit_LengthMovelist 
#define m_fit_NrRedundancyGridDepth 
#define m_fit_NrRedundancyAssemblies 
#define m_fit_TileIndexStartingTile 

#define mEMPTY_CELL 255
#define mEMPTY_CELL_ML 22<<2 //254
#define mEMPTY_CELL_OUT_OF_BOUNDS 253
//#include <stdio.h>




struct xThreadInfo {
        ushort4 data;
        
        __device__  xThreadInfo(unsigned short __usThreadIdX, unsigned short __usThreadIdY, unsigned short __usBlockIdX, unsigned short __usBlockIdY);
        __device__ unsigned short WarpId(void);
        __device__ unsigned short BankId(void);
        __device__ unsigned short FlatThreadId(void);
        __device__ unsigned short FlatBlockId(void);
        __device__ unsigned short GlobId(unsigned short __usTypeLength);
        __device__ void __DEBUG_CALL(void);
};

__device__ xThreadInfo::xThreadInfo(unsigned short __usThreadIdX, unsigned short __usThreadIdY, unsigned short __usBlockIdX, unsigned short __usBlockIdY) {
        this->data.z = threadIdx.y * m_fit_DimThreadX + threadIdx.x; //Flat Thread ID
        this->data.x = this->data.z % mWarpSize; //BankID
        this->data.y = (this->data.z - this->data.x) / mWarpSize; //WarpID
        this->data.w = blockIdx.y * m_fit_DimBlockX + blockIdx.x; //Flat Block ID
}

__device__ unsigned short xThreadInfo::WarpId(void) {
        return this->data.y;
}
__device__ unsigned short xThreadInfo::BankId(void) {
        return this->data.x;
}
__device__ unsigned short xThreadInfo::FlatThreadId(void) {
        return this->data.z;
}
__device__ unsigned short xThreadInfo::FlatBlockId(void) {
        return this->data.w;
}

__device__ unsigned short xThreadInfo::GlobId(unsigned short __usTypeLength) {
        return (this->data.w * m_fit_DimThreadX * m_fit_DimThreadY + this->data.z) * __usTypeLength;
}

struct xGenome {
        union{
            unsigned char one_d[mAlignedByteLengthGenome];
        } data;

        __device__ void CopyFromGlobal(xThreadInfo __xThreadInfo, unsigned char *__g_ucGenomeSet);
	__device__ void CopyToGlobal(xThreadInfo __xThreadInfo, unsigned char *__g_ucGenomeSet);
        __device__ unsigned char get_xEdgeType(unsigned char __ucTileId, unsigned char __ucEdgeId);
        __device__ void set_EdgeType(xThreadInfo *__xThreadInfo, unsigned char __ucTileId, unsigned char __ucEdgeId, unsigned char __ucVal);
};

struct xGenomeSet {
        union{        
            xGenome multi_d[mWarpSize];        
            unsigned char one_d[mWarpSize*sizeof(xGenome)];
        } data;

        __device__ void CopyFromGlobal(xThreadInfo __xThreadInfo, unsigned char *__g_ucGenomeSet);
        __device__ void CopyToGlobal(xThreadInfo __xThreadInfo, unsigned char *__g_ucGenomeSet);
        __device__ unsigned char get_xEdgeType(xThreadInfo __xThreadInfo, unsigned char __ucTileId, unsigned char __ucEdgeId);
        __device__ unsigned char set_EdgeType(xThreadInfo *__xThreadInfo, unsigned char __ucTileId, unsigned char __ucEdgeId, unsigned char __ucVal);
        __device__ void print(xThreadInfo *__xThreadInfo);
};

__device__ void xGenomeSet::CopyFromGlobal(xThreadInfo __xThreadInfo, unsigned char *__g_ucGenomes) {
        this->data.multi_d[__xThreadInfo.BankId()].CopyFromGlobal(__xThreadInfo, __g_ucGenomes);
}

__device__ void xGenome::CopyFromGlobal(xThreadInfo __xThreadInfo, unsigned char *__g_ucGenomes) {
        for (int i = 0; i < mAlignedByteLengthGenome; i ++) {
             this->data.one_d[i] = __g_ucGenomes[__xThreadInfo.GlobId(sizeof(xGenome)) + i];
        }
}

__device__ void xGenome::CopyToGlobal(xThreadInfo __xThreadInfo, unsigned char *__g_ucGenomes) {
        for (int i = 0; i < mAlignedByteLengthGenome; i += 1) {
                //(*reinterpret_cast<int*> (&this->data.one_d[i])) = (*reinterpret_cast<int*> (&__g_ucGenomeSet[__xThreadInfo->GlobId(sizeof(xGenome)) + i]));
                __g_ucGenomes[__xThreadInfo.GlobId(sizeof(xGenome)) + i] = this->data.one_d[i];
        }
}

__device__ void xGenomeSet::CopyToGlobal(xThreadInfo __xThreadInfo, unsigned char *__g_ucGenomes) {
        this->data.multi_d[__xThreadInfo.BankId()].CopyToGlobal(__xThreadInfo, __g_ucGenomes);
}

__device__ unsigned char xGenome::get_xEdgeType( unsigned char __ucTileId, unsigned char __ucEdgeId){
    if (__ucTileId < mNrTileTypes) {
        unsigned short TmpStartBit = __ucTileId * mBitLengthEdgeType * mNrTileOrientations + __ucEdgeId * mBitLengthEdgeType;
	unsigned short TmpEndBit = TmpStartBit + mBitLengthEdgeType;
	unsigned char TmpRetVal = 0;
	unsigned short TmpByteOffset = 0;
	unsigned short TmpBitOffset = 0;
	//Note: This could be speeded up by copying all bits within a byte simultaneously
	unsigned short j = 0;
	for (int i = TmpStartBit; i < TmpEndBit; i++) {
	    TmpBitOffset = i % 8; //We need to invert index as we start from left to right
	    TmpByteOffset = (i - TmpBitOffset) / 8;
	    TmpBitOffset = 7 - TmpBitOffset;
	    TmpRetVal += mOneOrNil(mBitTest(TmpBitOffset, this->data.one_d[TmpByteOffset])) << (mBitLengthEdgeType - 1 - j);
	    j++;
            //return mBitTest(TmpBitOffset,8);
	}
        return TmpRetVal;
    } else return (unsigned char) 0x00;
}

__device__ unsigned char xGenomeSet::get_xEdgeType(xThreadInfo __xThreadInfo, unsigned char __ucTileId, unsigned char __ucEdgeId){
    return this->data.multi_d[__xThreadInfo.BankId()].get_xEdgeType(__ucTileId, __ucEdgeId);
}

//GENOME TEST KERNEL
__global__ void TestGenomeKernel(unsigned char *dest)
{
    __shared__ xGenomeSet Tmp;
    xThreadInfo Tmpa(threadIdx.x, threadIdx.y, blockIdx.x, blockIdx.y);
    Tmp.CopyFromGlobal(Tmpa, dest);   
    for(int i=0;i<4;i++){
        Tmp.data.multi_d[Tmpa.BankId()].data.one_d[i] = Tmp.get_xEdgeType(Tmpa, i, 1);
    }
    Tmp.CopyToGlobal(Tmpa, dest);
}



texture<float, 2> t_ucInteractionMatrix;

struct xAssemblyFlags {
	unsigned char bitset;
	unsigned char bitset2;
	unsigned char red;
	unsigned char fullcheckcutoff;

	__device__ void set_Red(unsigned char __ucVal);
	__device__ void set_TrivialUND(void);
	__device__ void set_UnboundUND(void);
	__device__ void set_StericUND(void);
	__device__ void set_BusyFlag(void);
	__device__ bool get_bTrivialUND(void);
	__device__ bool get_bUnboundUND(void);
	__device__ bool get_bStericUND(void);
	__device__ bool get_bBusyFlag(void);
	__device__ bool get_bUNDCondition(void);
	__device__ unsigned char get_ucRed(void);
	__device__ void ClearAll(void);
	__device__ void ClearBitsets(void);
};

__device__ void xAssemblyFlags::set_Red(unsigned char __ucVal) {
    this->red = __ucVal;
}

__device__ void xAssemblyFlags::set_TrivialUND(void) {
    this->bitset |= (1 << 4);  //TEST
}

__device__ void xAssemblyFlags::set_UnboundUND(void) {
    this->bitset |= (1 << 5); //TEST
}

__device__ void xAssemblyFlags::set_StericUND(void) {
    this->bitset |= (1 << 6); //TEST
}

__device__ void xAssemblyFlags::set_BusyFlag(void) {
    this->bitset |= (1 << 7);
}

__device__ bool xAssemblyFlags::get_bTrivialUND(void) {
    return (bool) (this->bitset & (1 << 4));
}

__device__ bool xAssemblyFlags::get_bUnboundUND(void) {
    return (bool) (this->bitset & (1 << 5));
}

__device__ bool xAssemblyFlags::get_bStericUND(void) {
    return (bool) (this->bitset & (1 << 6));
}

__device__ bool xAssemblyFlags::get_bBusyFlag(void) {
    return (bool) (this->bitset & (1 << 7));
}

__device__ bool xAssemblyFlags::get_bUNDCondition(void) {
    return (bool) (this->bitset & 120);
}

__device__ unsigned char xAssemblyFlags::get_ucRed(void) {
    return this->red;
}

__device__ void xAssemblyFlags::ClearAll(void) {
    this->bitset = 0;
    this->bitset2 = 0;
    this->red=0;
    this->fullcheckcutoff=0;
    return;
}

__device__ void xAssemblyFlags::ClearBitsets(void) {
    this->bitset = 0;
    this->bitset2 = 0;
    return;
}


struct xEdgeSort {
	union {
		unsigned char multi_d[mNrEdgeTypes][mNrTileTypes][mNrTileOrientations][mWarpSize];
                unsigned char mix_d[mNrEdgeTypes*mNrTileTypes*mNrTileOrientations][mWarpSize];
		unsigned char one_d[mNrEdgeTypes * mNrTileTypes * mNrTileOrientations * mWarpSize];
	} data;

	union {
		unsigned short multi_d[mNrEdgeTypes][mWarpSize];
		unsigned char one_d[mNrEdgeTypes * mWarpSize * sizeof(short)];
	} length;

	__device__ void Zeroise(xThreadInfo __xThreadInfo);
        __device__ void Initialise(xThreadInfo __xThreadInfo, xGenomeSet *__xGenomeSet, short __sEdgeId = -1);
        __device__ unsigned char GetBondingTile(xThreadInfo __xThreadInfo, short __sEdgeId, hiprandState *__xCurandState, xAssemblyFlags *__xAssemblyFlags);
        __device__ void add_TileOrient(xThreadInfo __xThreadInfo, unsigned char __ucEdgeId, unsigned char __ucOrient, unsigned char __ucTileType);
	__device__ __forceinline__ void set_xLength(xThreadInfo __xThreadInfo, unsigned char __ucEdgeId, unsigned char __ucLength);
        __device__ void add_Tile(xThreadInfo __xThreadInfo, unsigned char __ucEdgeId);
        __device__ unsigned char get_xData(xThreadInfo __xThreadInfo, unsigned char __ucEdgeId, unsigned char __ucTileId, unsigned char __ucOrientation);
        __device__ unsigned char GetBondingTileOrientation(xThreadInfo __xThreadInfo, unsigned char __ucEdgeId, unsigned char __ucTileId, xAssemblyFlags *__xAssemblyFlags);
        __device__ short get_xLength(xThreadInfo __xThreadInfo, unsigned short __sEdgeId);
};

#define mDelta(a,b) ((a==b)?1:0)
__device__ __forceinline__ int InteractionMatrix(int i, int j){
	return (1-i%2)*mDelta(i,j+1)+(i%2)*mDelta(i,j-1);
}

__device__ void xEdgeSort::Initialise(xThreadInfo __xThreadInfo, xGenomeSet *__xGenomeSet, short __sEdgeId) {
          
        //DEBUG
        /*for(int i=0; i< mNrEdgeTypes*mNrTileTypes*mNrTileOrientations;i++){
           
        }*/
        //DEBUG

	//if (__sEdgeId == -1) {
/*		for (int k = 0; k < mNrEdgeTypes; k++) {
                //if(__xThreadInfo.WarpId() < mNrEdgeTypes){
                //int k = __xThreadInfo.WarpId();
			this->set_xLength(__xThreadInfo, k, 0);
			bool r_bEdgeAdded = false;
			for (int i = 0; i < mNrTileTypes; i++) { //Traverse all TileTypes
				for (int j = 0; j < mNrTileOrientations; j++) { //Traverse all Orientations
					//if (tex2D(t_ucInteractionMatrix,
				        //		__xGenomeSet->get_xEdgeType(__xThreadInfo, i, j), k)
					//		> 0) { //Does Edge j of Tile i bond to Tile ThreadID
                                        if(InteractionMatrix(__xGenomeSet->get_xEdgeType(__xThreadInfo, i, j), k)){
						this->add_TileOrient(__xThreadInfo, k, j, i);
						r_bEdgeAdded = true;
					} else {
						this->add_TileOrient(__xThreadInfo, k, j, mEMPTY_CELL);
					}
				}
				if (r_bEdgeAdded) { //EdgeAdded?
					this->add_Tile(__xThreadInfo, k); //TEST
					r_bEdgeAdded = 0;
				} else {
					//Do Nothing
				}
			}
                       //set_xLength(__xThreadInfo, 0, 5); //Test: 
		}
*/
               if(__xThreadInfo.WarpId()==0){
               //if(threadIdx.x==0){

               for (int k = 0; k < mNrEdgeTypes; k++) {
			this->set_xLength(__xThreadInfo, k, 0);
			bool r_bEdgeAdded = false;
			for (int i = 0; i < mNrTileTypes; i++) { /*Traverse all TileTypes*/
				for (int j = 0; j < mNrTileOrientations; j++) { /*Traverse all Orientations*/
               //                         printf("%d|", __xGenomeSet->get_xEdgeType(__xThreadInfo, i, j));
					if (tex2D(t_ucInteractionMatrix,
							__xGenomeSet->get_xEdgeType(__xThreadInfo, i, j), k)
							> 0) { /*Does Edge j of Tile i bond to Tile ThreadID*/
						this->add_TileOrient(__xThreadInfo, k, j, i);
						r_bEdgeAdded = true;
					} else {
						this->add_TileOrient(__xThreadInfo, k, j, mEMPTY_CELL);
					}
				}
				if (r_bEdgeAdded) { /*EdgeAdded?*/
					this->add_Tile(__xThreadInfo, k); //TEST
					r_bEdgeAdded = 0;
				} else {
					/*Do Nothing*/
				}
                //                printf ("\n");
			}
		}
        //        printf("Next one...")
                }

	/*} else {
		this->set_xLength(__xThreadInfo, __sEdgeId, 0);
		bool r_bEdgeAdded = false;
		for (int i = 0; i < mNrTileTypes; i++) { //Traverse all TileTypes
			for (int j = 0; j < mNrTileOrientations; j++) {//Traverse all Orientations
				if (tex2D(t_ucInteractionMatrix, __xGenomeSet->get_xEdgeType(
						__xThreadInfo, i, j), __sEdgeId) > 0) { //Does Edge j of Tile i bond to Tile ThreadID
                                //if(InteractionMatrix(__xGenomeSet->get_xEdgeType(__xThreadInfo, i, j), k)){
					this->add_TileOrient(__xThreadInfo, __sEdgeId, j, i);
					r_bEdgeAdded = true;
				} else {
					this->add_TileOrient(__xThreadInfo, __sEdgeId, j,
							mEMPTY_CELL);
				}
			}
			if (r_bEdgeAdded) { //EdgeAdded?
				this->add_Tile(__xThreadInfo, __sEdgeId);
				r_bEdgeAdded = 0;
			} else {
				//Do Nothing
			}
		}
	}*/
}

__device__ __forceinline__ void xEdgeSort::set_xLength(xThreadInfo __xThreadInfo, unsigned char __ucEdgeId, unsigned char __ucLength) {
	this->length.multi_d[__ucEdgeId][__xThreadInfo.BankId()] = __ucLength;
}

__device__ void xEdgeSort::add_TileOrient(xThreadInfo __xThreadInfo, unsigned char __ucEdgeId, unsigned char __ucOrient, unsigned char __ucTileType) {
	this->data.multi_d[__ucEdgeId][this->get_xLength(__xThreadInfo, __ucEdgeId)][__ucOrient][__xThreadInfo.BankId()] = __ucTileType;
}

__device__ short xEdgeSort::get_xLength(xThreadInfo __xThreadInfo, unsigned short __sEdgeId) {
	if (__sEdgeId < mNrEdgeTypes) {
		return this->length.multi_d[__sEdgeId][__xThreadInfo.BankId()];
	} else {
		return 0;
	}
}

void xEdgeSort::add_Tile(xThreadInfo __xThreadInfo, unsigned char __ucEdgeId) {
	this->set_xLength(__xThreadInfo, __ucEdgeId, this->get_xLength(	__xThreadInfo, __ucEdgeId) + 1);
}

__global__ void TestEdgeSortKernel(unsigned char *dest, hiprandState *states)
{
    __shared__ xGenomeSet s_xGenomeSet;
    __shared__ xEdgeSort s_xEdgeSort;
    xThreadInfo r_xThreadInfo(threadIdx.x, threadIdx.y, blockIdx.x, blockIdx.y);
    s_xGenomeSet.CopyFromGlobal(r_xThreadInfo, dest);
    s_xEdgeSort.Initialise(r_xThreadInfo, &s_xGenomeSet, -1); 
    for(int i=0;i<4;i++){
        //s_xGenomeSet.data.multi_d[r_xThreadInfo.BankId()].data.one_d[i] = s_xEdgeSort.length.multi_d[i][r_xThreadInfo.BankId()];
        s_xGenomeSet.data.multi_d[r_xThreadInfo.BankId()].data.one_d[i] = s_xEdgeSort.data.multi_d[6][0][i][r_xThreadInfo.BankId()];
        //s_xGenomeSet.data.multi_d[r_xThreadInfo.BankId()].data.one_d[i] = tex2D(t_ucInteractionMatrix, i, 1);
    }
    s_xGenomeSet.CopyToGlobal(r_xThreadInfo, dest);
}













#include <stdio.h>

#define m_fit_LengthMovelist 20
#define mEMPTY_CELL 255




extern "C++"{

template<class T>
struct xLifoList {
        struct {
                signed short pos; //Current position of top element (-1...max_length-1)
        } data;

        __device__ bool bPush(xThreadInfo __xThreadInfo, T __xEntry, T (&__xStorage)[m_fit_LengthMovelist][mWarpSize], unsigned short __uiMaxLength);
	__device__ T xPop(xThreadInfo __xThreadInfo, T (&__xStorage)[m_fit_LengthMovelist][mWarpSize]);
        __device__ short get_sPos();
        __device__ short set_sPos(short __sPos);
};

template<class T>
__device__ bool xLifoList<T>::bPush(xThreadInfo __xThreadInfo, T __xEntry, T (&__xStorage)[m_fit_LengthMovelist][mWarpSize], unsigned short __uiMaxLength) {
        if (this->data.pos < __uiMaxLength) {
                __xStorage[this->data.pos][__xThreadInfo.BankId()] = __xEntry;
                this->data.pos++;
                return true;
        } else {
                return false;
        }
}

template<class T>
__device__ T xLifoList<T>::xPop(xThreadInfo __xThreadInfo, T (&__xStorage)[m_fit_LengthMovelist][mWarpSize]) {
        if (this->data.pos <= 0) { //FXD
                //NOTE: ADAPTED FOR CUDA VECTORTYPES ONLY!
                T buf;
                buf.x = mEMPTY_CELL;
                buf.y = mEMPTY_CELL;
                return buf;
        } else {
                this->data.pos--;
                return __xStorage[this->data.pos][__xThreadInfo.BankId()];
        }
}

template<class T>
__device__ short xLifoList<T>::get_sPos() {
        return this->data.pos;
}

template<class T>
__device__ short xLifoList<T>::set_sPos(short __sPos){
        this->data.pos = __sPos;
        return 0;
}

template<class T>
struct xMoveList {
        struct {
                T multi_d[m_fit_LengthMovelist][mWarpSize];
        } storage;
        struct {
                xLifoList<T> multi_d[mWarpSize];
        } list;

        __device__ void Initialise(xThreadInfo __xThreadInfo);
        __device__ bool bPush(xThreadInfo __xThreadInfo, T __xEntry);
        __device__ T xPop(xThreadInfo __xThreadInfo);
        __device__ short get_sPos(xThreadInfo __xThreadInfo);
        __device__ short set_sPos(xThreadInfo __xThreadInfo, short __sPos);
};

template<class T>
__device__ bool xMoveList<T>::bPush(xThreadInfo __xThreadInfo, T __xEntry) {
        this->list.multi_d[__xThreadInfo.BankId()].bPush(__xThreadInfo, __xEntry, this->storage.multi_d, );
        return true;
}

template<class T>
__device__ void xMoveList<T>::Initialise(xThreadInfo __xThreadInfo) {
        this->list.multi_d[__xThreadInfo.BankId()].data.pos = 0;
}

template<class T>
__device__ T xMoveList<T>::xPop(xThreadInfo __xThreadInfo) {
	return this->list.multi_d[__xThreadInfo.BankId()].xPop(__xThreadInfo, this->storage.multi_d);
}

template<class T>
__device__ short xMoveList<T>::get_sPos(xThreadInfo __xThreadInfo) {
        return this->list.multi_d[__xThreadInfo.BankId()].get_sPos();
}

template<class T>
__device__ short xMoveList<T>::set_sPos(xThreadInfo __xThreadInfo, short __sPos){
        return this->list.multi_d[__xThreadInfo.BankId()].set_sPos(__sPos);
}

}

//MOVELIST TEST KERNEL
__global__ void TestMovelistKernel(unsigned char *dest)
{
    /*__shared__ xGenomeSet Tmp;
    xThreadInfo Tmpa(threadIdx.x, threadIdx.y, blockIdx.x, blockIdx.y);
    Tmp.CopyFromGlobal(Tmpa, dest);   
    for(int i=0;i<4;i++){
        Tmp.data.multi_d[Tmpa.BankId()].data.one_d[i] = Tmp.get_xEdgeType(Tmpa, i, 1);
    }
    Tmp.CopyToGlobal(Tmpa, dest);*/

    xThreadInfo __xThreadInfo(threadIdx.x, threadIdx.y, blockIdx.x, blockIdx.y); 
    xMoveList<uchar2> s_xMovelist;
    s_xMovelist.Initialise(__xThreadInfo);
    s_xMovelist.bPush(__xThreadInfo, make_uchar2(9,9));
    s_xMovelist.bPush(__xThreadInfo, make_uchar2(6,6));
    for(int i=0;i<2;i++){
        //dest[__xThreadInfo.BankId()*2 + i] = s_xMovelist.storage.multi_d[0][__xThreadInfo.BankId()].x;//s_xMovelist.xPop(__xThreadInfo).x;
        //dest[__xThreadInfo.BankId()*(s_xMovelist.get_sPos(__xThreadInfo)+1) + i] = s_xMovelist.xPop(__xThreadInfo).x;
        //dest[__xThreadInfo.BankId()*(s_xMovelist.get_sPos(__xThreadInfo)+1) + i] = ;
    }
}

__constant__ unsigned char c_ucFourPermutations[][];

struct xFourPermutation {
	unsigned short WalkIndex;
        uchar4 Perm;
	__device__ xFourPermutation(unsigned short __usPermIndex);
	__device__ unsigned short ucWalk();
	__device__ bool bNotTraversed();
};

__device__ xFourPermutation::xFourPermutation(unsigned short __usPermIndex) {
    this->WalkIndex = 0;
    switch(__usPermIndex % 24){
        case 1: this->Perm = make_uchar4(1,2,3,4); break;
        case 2: this->Perm = make_uchar4(1,2,4,3); break;
        case 3: this->Perm = make_uchar4(1,3,2,4); break;
        case 4: this->Perm = make_uchar4(1,3,4,2); break;
        case 5: this->Perm = make_uchar4(1,4,2,3); break;
        case 6: this->Perm = make_uchar4(1,4,3,2); break;
        case 7: this->Perm = make_uchar4(2,1,3,4); break;
        case 8: this->Perm = make_uchar4(2,1,4,3); break;
        case 9: this->Perm = make_uchar4(2,3,1,4); break;
        case 10: this->Perm = make_uchar4(2,3,4,1); break;
        case 11: this->Perm = make_uchar4(2,4,1,3); break;
        case 12: this->Perm = make_uchar4(2,4,3,1); break;
        case 13: this->Perm = make_uchar4(3,2,1,4); break;
        case 14: this->Perm = make_uchar4(3,2,4,1); break;
        case 15: this->Perm = make_uchar4(3,1,2,4); break;
        case 16: this->Perm = make_uchar4(3,1,4,2); break;
        case 17: this->Perm = make_uchar4(3,4,2,1); break;
        case 18: this->Perm = make_uchar4(3,4,1,2); break;
        case 19: this->Perm = make_uchar4(4,2,3,1); break;
        case 20: this->Perm = make_uchar4(4,2,1,3); break;
        case 21: this->Perm = make_uchar4(4,3,2,1); break;
        case 22: this->Perm = make_uchar4(4,3,1,2); break;
        case 23: this->Perm = make_uchar4(4,1,2,3); break;
        case 0: this->Perm = make_uchar4(4,1,3,2); break;
    }
}

__device__ unsigned short xFourPermutation::ucWalk() {
    //Require c_ucFourPermutations to be numbers 1-4 (NOT 0-3)
    this->WalkIndex++;
    if (this->WalkIndex - 1 < mNrTileOrientations) {
        //return this->Perm[];//this->WalkIndex-1; //c_ucFourPermutations[this->PermIndex][this->WalkIndex - 1] - 1; //TEST
        switch(this->WalkIndex-1){
            case 0: return this->Perm.x-1;
            case 1: return this->Perm.y-1;
            case 2: return this->Perm.z-1;
            case 3: return this->Perm.w-1;            
        }
    } else return 0;
}

__device__ bool xFourPermutation::bNotTraversed() {
    //Require c_ucFourPermutations to be numbers 1-4 (NOT 0-3)
    if (this->WalkIndex >= mNrTileOrientations) {
        return false;
    } else return true;
}

extern "C++"{
template<int Length>
struct xLinearIterator {
	unsigned short WalkIndex;
	__device__ xLinearIterator(unsigned short __usPermIndex);
	__device__ unsigned short ucWalk();
	__device__ bool bNotTraversed();
};

template<int Length>
__device__ xLinearIterator<Length>::xLinearIterator(unsigned short __usPermIndex) {
    //this->WalkIndex = 0;
}

template<int Length>
__device__ unsigned short xLinearIterator<Length>::ucWalk() {
    //Require c_fFourPermutations to be numbers 1-4 (NOT 0-3)
    this->WalkIndex++;
    if (this->WalkIndex - 1 < Length) {
        return this->WalkIndex - 1;
    } else return 0;
}

template<int Length>        
__device__ bool xLinearIterator<Length>::bNotTraversed() {
    //Require c_fFourPermutations to be numbers 1-4 (NOT 0-3)
    if (this->WalkIndex >= Length) {
        return false;
    } else return true;
}

struct xCell {
	unsigned char data;
        __device__ void set_Orient(unsigned char __uiOrient);
        __device__ void set_Type(unsigned char __uiType);
        __device__ unsigned char get_xType(void);
        __device__ unsigned char get_xOrient(void);
        __device__ unsigned char get_xCell(void);
        __device__ void set_xCell(unsigned char __ucVal);
};

__device__ void xCell::set_Orient(unsigned char __uiOrient) {
	__uiOrient = __uiOrient % mNrTileOrientations;
	//unsigned char DBGVAL1 = this->data & (255-3);
	//unsigned char DBGVAL2 = __uiOrient;
	//unsigned char DBGVAL3 = this->data & (255-3) + __uiOrient;
	//I THINK THIS FUNCTION DOES NOT WORK!
	this->data = ((this->data & (255-3) ) + __uiOrient);
}

__device__ void xCell::set_Type(unsigned char __uiType) {
#ifndef __NON_FERMI
	if (__uiType > 63) {
		printf("xCell: TileType exceeded 63 limit!\n");
	}
#endif
	this->data = (this->data & 3) + (__uiType << 2);
}

__device__ void xCell::set_xCell(unsigned char __ucVal) {
	this->data = __ucVal;
}

__device__ unsigned char xCell::get_xType(void) {
	return this->data >> 2;
}

__device__ unsigned char xCell::get_xOrient(void) {
	return (this->data & 3);
}

__device__ unsigned char xCell::get_xCell(void) {
	return this->data;
}

struct xCellGrid {
	union {
		xCell multi_d[m_fit_DimGridX][m_fit_DimGridY][m_fit_NrRedundancyGridDepth][mWarpSize];
		xCell mix_d[m_fit_DimGridX * m_fit_DimGridY][m_fit_NrRedundancyGridDepth][mWarpSize];
		xCell one_d[m_fit_DimGridX * m_fit_DimGridY * mWarpSize	* m_fit_NrRedundancyGridDepth];
	} data;

	__device__ void Initialise(xThreadInfo __xThreadInfo, unsigned char __red);
        __device__ xCell get_xCell(xThreadInfo __xThreadInfo, unsigned char __x, unsigned char __y, unsigned char __red);
        __device__ bool set_xCell(xThreadInfo __xThreadInfo, unsigned char __x, unsigned char __y, unsigned char __red, unsigned char __val);
        __device__ xCell xGetNeighbourCell(xThreadInfo __xThreadInfo, unsigned char __x, unsigned char __y, unsigned char __red, unsigned char __dir);
        __device__ uchar2 xGetNeighbourCellCoords(unsigned char __x, unsigned char __y, unsigned char __dir);
        __device__ bool xCompareRed(xThreadInfo __xThreadInfo, unsigned char __red);
        __device__ void print(xThreadInfo __xThreadInfo, xGenomeSet *__xGenomeSet);
};

__device__ void xCellGrid::Initialise(xThreadInfo __xThreadInfo,
		unsigned char __red) {
	//Surefire-version:
	/*for (int i = 0; i < m_fit_DimGridX; i++) {
		for (int j = 0; j < m_fit_DimGridY; j++) {
			this->data.multi_d[i][j][__red][__xThreadInfo.BankId()].set_xCell(mEMPTY_CELL);
		}
	}*/
        /*for (int i = 0; i < m_fit_DimGridX; i++) {
                for (int j = 0; j < m_fit_DimGridY; j++) {
                        this->data.multi_d[i*j][__red][__xThreadInfo.BankId()].set_xCell(mEMPTY_CELL);
                }
        }*/
        short offset = (m_fit_DimGridX*m_fit_DimGridY) % mBankSize;
        short myshare = (m_fit_DimGridX*m_fit_DimGridY - offset) / mBankSize; 

        for(int i=0;i<myshare;i++){
                this->data.mix_d[__xThreadInfo.WarpId()*myshare + i][__red][__xThreadInfo.BankId()].set_xCell(mEMPTY_CELL); 
        }
        if(__xThreadInfo.WarpId()==mBankSize-1){
                for(int i=0;i<offset;i++){
                        this->data.mix_d[mBankSize*myshare + i][__red][__xThreadInfo.BankId()].set_xCell(mEMPTY_CELL);
                }
        }
}

__device__ xCell xCellGrid::get_xCell(xThreadInfo __xThreadInfo, unsigned char __x, unsigned char __y, unsigned char __red) {
        if ( (__x <= m_fit_DimGridX-1) && (__x >= 0) && (__y <= m_fit_DimGridY-1) && (__y >= 0) ) { // In grid
            return this->data.multi_d[__x][__y][__red%m_fit_NrRedundancyGridDepth][__xThreadInfo.BankId()];
        } else { // Outside of grid
            xCell TmpCell;
            TmpCell.set_xCell(mEMPTY_CELL_OUT_OF_BOUNDS);
            return TmpCell;
        }
}

__device__ bool xCellGrid::set_xCell(xThreadInfo __xThreadInfo, unsigned char __x, unsigned char __y, unsigned char __red, unsigned char __val) {
        if ( (__x <= m_fit_DimGridX-1) && (__x >= 0) && (__y <= m_fit_DimGridY-1) && (__y >= 0) ) { // In grid
            this->data.multi_d[__x][__y][__red%m_fit_NrRedundancyGridDepth][__xThreadInfo.BankId()].set_xCell(__val);
        } 

        if ( (__x >= m_fit_DimGridX-1) || (__x <= 0) ||  (__y >= m_fit_DimGridY-1) || (__y <= 0) ) { // In grid
            return false;
        } else return true;
}

__device__ xCell xCellGrid::xGetNeighbourCell(xThreadInfo __xThreadInfo, unsigned char __x, unsigned char __y, unsigned char __red, unsigned char __dir) {
	uchar2 TmpCoords = xGetNeighbourCellCoords(__x, __y, __dir);
	return this->get_xCell(__xThreadInfo, TmpCoords.x, TmpCoords.y, __red);
}

__device__ uchar2 xCellGrid::xGetNeighbourCellCoords(unsigned char __x, unsigned char __y, unsigned char __dir) {
	switch (__dir) {
        case 0: //NORTH
                return make_uchar2(__x, __y - 1);
                //break;
	case 1: //EAST
		return make_uchar2(__x + 1, __y);
		//break;
        case 2: //SOUTH
                return make_uchar2(__x, __y + 1);
                //break;
	case 3: //WEST
		return make_uchar2(__x - 1, __y);
		//break;
	}
	return make_uchar2(mEMPTY_CELL, mEMPTY_CELL);
}

__device__ bool xCellGrid::xCompareRed(xThreadInfo __xThreadInfo, unsigned char __red) {
        unsigned char TmpNextDir = (__red + 1) % m_fit_NrRedundancyGridDepth;
	unsigned char TmpIsDifferent = 0;
	for (int i = 0; i < m_fit_DimGridX * m_fit_DimGridY; i++) {
		if (this->data.mix_d[i][__red][__xThreadInfo.BankId()].get_xCell() != this->data.mix_d[i][TmpNextDir][__xThreadInfo.BankId()].get_xCell() ) {
		    TmpIsDifferent = 1;
		    break;
		}
	}
	if (!TmpIsDifferent)
		return true;
	else
		return false;
}

struct xFitnessGrid {
	texture<xCell, 2> *grid;
	__device__ unsigned char get_xCell(unsigned char i, unsigned char j);
};

struct xAssembly {
	struct {
		xCellGrid grid;
		xEdgeSort edgesort;
		xMoveList<uchar2> movelist;
		xAssemblyFlags flags[mWarpSize];
		hiprandState *states;
                int assembly_size[mWarpSize];
                int4 hash[mWarpSize];
                int2 corner_lower[mWarpSize];
                int2 corner_upper[mWarpSize];
	} data;

	__device__ void Initialise(xThreadInfo __xThreadInfo, xGenomeSet *__xGenomeSet);
	__device__ bool Assemble(xThreadInfo __xThreadInfo, xGenomeSet *__xGenomeSet);
        __device__ bool Assemble_PreProcess(xThreadInfo __xThreadInfo, xGenomeSet *__xGenomeSet);
	__device__ bool Assemble_PostProcess(xThreadInfo __xThreadInfo, xGenomeSet *__xGenomeSet);
	__device__ bool Assemble_Movelist(xThreadInfo __xThreadInfo, xGenomeSet *__xGenomeSet);
	__device__ bool Assemble_InPlace(xThreadInfo __xThreadInfo, xGenomeSet *__xGenomeSet);
	__device__ float fEvaluateFitness(xThreadInfo __xThreadInfo);
	__device__ bool bSynchronizeBank(xThreadInfo __xThreadInfo);
};

__device__ void jenkins_init(int &hash){
    hash = 0;
}

__device__ void jenkins_add(char key, int &hash){
    unsigned int tmphash=hash;
    tmphash += key;
    tmphash += (tmphash << 10);
    tmphash ^= (tmphash >> 6);
    hash=tmphash;
}

__device__ unsigned int jenkins_clean_up(int &hash){
    unsigned int tmphash=hash;
    tmphash += (tmphash << 3);
    tmphash ^= (tmphash >> 11);
    tmphash += (tmphash << 15);
    hash = tmphash;
    return hash;
}


__device__ void xAssembly::Initialise(xThreadInfo __xThreadInfo, xGenomeSet *__xGenomeSet) {
	unsigned char TmpRed = this->data.flags[__xThreadInfo.BankId()].get_ucRed() % m_fit_NrRedundancyGridDepth;
	this->data.grid.Initialise(__xThreadInfo, TmpRed);
	this->data.movelist.Initialise(__xThreadInfo);
}

__device__ bool xAssembly::Assemble(xThreadInfo __xThreadInfo, xGenomeSet *__xGenomeSet) {
	bool TmpFlag = false;
	this->data.flags[__xThreadInfo.BankId()].ClearAll();
	TmpFlag = true; 
	if (TmpFlag) {
                if(__xThreadInfo.WarpId() == 0){ //DEBUG
              		this->data.edgesort.Initialise(__xThreadInfo, __xGenomeSet, -1); //TEST
                }
                __syncthreads();
		//this->Assemble_PostProcess(__xThreadInfo, __xGenomeSet);
		if (TmpFlag) {
			//for (int i = 0; (i < m_fit_NrRedundancyAssemblies) && (!this->data.flags[__xThreadInfo.BankId()].get_bUNDCondition()); i++) {
                        for (int i = 0; (i < m_fit_NrRedundancyAssemblies); i++) {
                                //__syncthreads();
                                if(!this->data.flags[__xThreadInfo.BankId()].get_bUNDCondition()){
					this->Initialise(__xThreadInfo, __xGenomeSet); //Empty out assembly grid at red
                                }
                                __syncthreads();
                                if( (__xThreadInfo.WarpId() == 0) && (!this->data.flags[__xThreadInfo.BankId()].get_bUNDCondition()) ){
                                        this->data.flags[__xThreadInfo.BankId()].ClearAll();
             			        this->Assemble_Movelist(__xThreadInfo, __xGenomeSet); //TEST
                                }
				//this->data.flags[__xThreadInfo.BankId()].set_Red(i); //Choose next assembly step!
                                return true; //DEBUG
			}
			return true; //Always true - i.e. indicate assembly did finish (can still be UND, though)
		} else {
			return false; //Indicates that processing before assembly returned either single block, or UND
		}

	} else {
		return false; //Indicates that processing before assembly returned either single block, or UND
	}

}

__device__ bool xAssembly::Assemble_PreProcess(xThreadInfo __xThreadInfo, xGenomeSet *__xGenomeSet) {
	unsigned char TmpSameCounter = 0;

	//NOTE: This should work, however, not clear how to communicate that single tile without initialisation of grid!
	//Check if starting tile is not empty
	for (int j = 0; j < mNrTileOrientations; j++) {
		if (__xGenomeSet->get_xEdgeType(__xThreadInfo, m_fit_TileIndexStartingTile,
				j) == 0)
			TmpSameCounter++;
	}
	if (TmpSameCounter == 4) {
		this->data.grid.get_xCell(__xThreadInfo, m_fit_DimGridX / 2,
				m_fit_DimGridY / 2, 0);
		return true; //Have finished assembly - UND is false, but so is PreProcess (trigger)
	}

	//Replace tile doublettes by empty tiles
	//Works for any number of mNrTileOrientations and mBitLengthEdgeType <= 4 Byte!
	//Note: This would be faster (but more inflexible) if tile-wise accesses!
	TmpSameCounter = 0;
	unsigned char DBGVAL1, DBGVAL2, DBGVAL3;
	for (int k = 0; k < mNrTileTypes - 1; k++) { //Go through all Tiles X (except for last one)
		for (int i = k + 1; i < mNrTileTypes; i++) { //Go through all Tiles X_r to the right
			for (int j = 0; j < mNrTileOrientations; j++) { //Go through all X edges rots
				TmpSameCounter = 0;
				for (int l = 0; l < mNrTileOrientations; l++) { //Cycle through all X edges
					if (__xGenomeSet->get_xEdgeType(__xThreadInfo, k, l)
							== __xGenomeSet->get_xEdgeType(__xThreadInfo, i, (j
									+ l) % mNrTileOrientations)) {
						TmpSameCounter++;
					}
				}
				if (TmpSameCounter == mNrTileOrientations) {
					//Have detected a doublette - replace with empty tile!!
					for (int l = 0; l < mNrTileOrientations; l++) {
						//__xGenomeSet->set_EdgeType(__xThreadInfo, i, l, 0); //TEST
					}
				}
			}
		}
	}
	return true;
}

__device__ bool xAssembly::Assemble_PostProcess(xThreadInfo __xThreadInfo, xGenomeSet *__xGenomeSet) {
	//Optional: start at first tile and see if it can connect to any degenerate entries in EdgeSort directly
	//Note: If we can refrain from assembly, then save time for grid initialisation!
	unsigned char TmpBondingCounter = 0;
	unsigned char TmpEdgeTypeLength = 0;
	for (int j = 0; j < mNrTileOrientations; j++) {
		TmpEdgeTypeLength = this->data.edgesort.get_xLength(__xThreadInfo, j);
		if (TmpEdgeTypeLength > 1) {
			this->data.flags[__xThreadInfo.BankId()].set_TrivialUND(); //TEST
			return false;
		} else if (TmpEdgeTypeLength == 0) {
			TmpBondingCounter++;
		}
	}

	if (TmpBondingCounter == 4) {
		//(Single-tile assembly: PostProcess return value is false, but UND is also false (trigger) )
		this->data.grid.set_xCell(__xThreadInfo, m_fit_DimGridX / 2, m_fit_DimGridY / 2, 0, 0);
		return false;
	}
	//Note: (Optional) Could now check for periodicity (can return to tile X first tile starting at X at same orientation)
	//Note: (Optional) Could now check for 2x2 assembly, etc (quite rare though)
	//NOTE: TODO, have to check in EdgeSort whether Tile is symmetric, i.e. then remove bonding orientations
	return true;
}

__device__ bool xAssembly::Assemble_Movelist(xThreadInfo __xThreadInfo, xGenomeSet *__xGenomeSet) {
	//Place tiletype 0 on center of grid
	this->data.grid.set_xCell(__xThreadInfo, m_fit_DimGridX / 2, m_fit_DimGridY / 2, this->data.flags[__xThreadInfo.BankId()].get_ucRed(), 0);
	//Add first four moves to movelist (even iff they might be empty)
	uchar2 X; //X be current position in grid
	X.x = m_fit_DimGridX / 2; //X is abused here as a buffer (reset at loop head)
	X.y = m_fit_DimGridY / 2;
        //return false; //TEST

        { //Keep all this local
	xFourPermutation BUF((int) (hiprand_uniform(&this->data.states[__xThreadInfo.BankId()])*24.0f));
	unsigned char index;
	while (BUF.bNotTraversed()) {
		index = BUF.ucWalk();
		//unsigned char DBGVAL = TmpAddPerm.WalkIndex;
                this->data.movelist.bPush(__xThreadInfo, this->data.grid.xGetNeighbourCellCoords(X.x, X.y, (unsigned char) (index)));
                this->data.grid.set_xCell( __xThreadInfo, this->data.grid.xGetNeighbourCellCoords(X.x, X.y, (unsigned char) (index)).x, this->data.grid.xGetNeighbourCellCoords(X.x, X.y, (unsigned char) (index)).y, this->data.flags[__xThreadInfo.BankId()].get_ucRed(), mEMPTY_CELL_ML);
                //this->data.grid.set_xCell( __xThreadInfo, 0, 0, this->data.flags[__xThreadInfo.BankId()].get_ucRed(), 250); //TEST
	}
        }
 
        //return false;
        //this->data.assembly_size[__xThreadInfo.BankId()] = 1;

        //BEGIN DEBUG
        /*for(int i=0;i<mNrEdgeTypes;i++){
        	for(int j=0;j<this->data.edgesort.get_xLength(__xThreadInfo, i);j++){
                       for(int k=0;k<4;k++){
	 	               this->data.grid.set_xCell( __xThreadInfo, j*4+k, i, this->data.flags[__xThreadInfo.BankId()].get_ucRed(), this->data.edgesort.data.multi_d[i][j][k][__xThreadInfo.BankId()]);
			}
                }
                this->data.grid.set_xCell( __xThreadInfo, this->data.edgesort.get_xLength(__xThreadInfo, i)*4, i, this->data.flags[__xThreadInfo.BankId()].get_ucRed(), 33<<2);
        } 
        return;*/
        //END DEBUG
 
	//We use movelist approach to assemble grids
	//Will switch to in-place assembly if either movelist full, or some other pre-defined condition.
	//Note: If we want mixed redundancy detection, need to implement some Single-Assembly Flag in AssemblyFlags that will switch.
	//Also: SynchronizeBank() needs to be adapted to not wait for other threads iff Many-thread approach!

	xCell N; //N(E_X) be non-empty neighbouring cells
	unsigned char Mirr; // Mirr(E_X, N(E_X)) be tile edge neighbouring E_X
	xCell T, TmpT; // T(Mirr(E_X, N(E_X)) be potential bonding tiles

        //BEGIN DEBUG
        //int DBG_MAXREP = 355;
        //int DBG_COUNTER = 0;
        //END DEBUG
	
        //For all elements M in Movelist (and while not UND condition detected)
 	while (this->data.movelist.get_sPos(__xThreadInfo) > 0) {
                //BEGIN DEBUG 
                //if(DBG_COUNTER >= DBG_MAXREP) return;
                //if(this->data.flags[__xThreadInfo.BankId()].get_bUNDCondition()){
                //	return;
                //}
                //STOP DEBUG

		//Choose position X from Movelist and remove it from Movelist
		X = this->data.movelist.xPop(__xThreadInfo);
                //Now remove grid marking to indicate movelist has been traversing this entry
                //this->data.grid.set_xCell( __xThreadInfo, X.x, X.y, this->data.flags[__xThreadInfo.BankId()].get_ucRed(), mEMPTY_CELL);

		T.set_xCell(mEMPTY_CELL);
                TmpT.set_xCell(mEMPTY_CELL);
		for (int E_X = 0; E_X < mNrTileOrientations; E_X++) {
			//BEGIN DEBUG
                        //this->data.grid.set_xCell( __xThreadInfo, 0, 0, this->data.flags[__xThreadInfo.BankId()].get_ucRed(), 33<<2);
                        //END DEBUG

			//::Let N(E_X) be non-empty neighbouring cells.
			N = this->data.grid.xGetNeighbourCell(__xThreadInfo, X.x, X.y, this->data.flags[__xThreadInfo.BankId()].get_ucRed(), (unsigned char) E_X);

                        //BEGIN DEBUG
                        //this->data.grid.set_xCell( __xThreadInfo, E_X, 0, this->data.flags[__xThreadInfo.BankId()].get_ucRed(), N.get_xCell());
                        //END DEBUG

			if ( (N.get_xCell() != mEMPTY_CELL) && (N.get_xCell() != mEMPTY_CELL_ML) && (N.get_xCell() != mEMPTY_CELL_OUT_OF_BOUNDS) ) { //For all N(E_X)
				//::Let Mirr(E_X, N(E_X)) be tile neighbouring E_X
				unsigned char TmpMirrorCoord = (4 - N.get_xOrient() + (E_X + mNrTileOrientations / 2) % mNrTileOrientations) % mNrTileOrientations;
				Mirr = __xGenomeSet->get_xEdgeType(__xThreadInfo, N.get_xType(), TmpMirrorCoord);
				//For all Mirr(E_X, N(E_X)), let T(Mirr(E_X, N(E_X)) be potential bonding tiles
				TmpT.set_xCell(this->data.edgesort.GetBondingTile( __xThreadInfo, Mirr, &this->data.states[__xThreadInfo.BankId()], &this->data.flags[__xThreadInfo.BankId()]));

				//BEGIN DEBUG
                                /*this->data.grid.set_xCell( __xThreadInfo, E_X, 1, this->data.flags[__xThreadInfo.BankId()].get_ucRed(), Mirr);
				this->data.grid.set_xCell( __xThreadInfo, E_X, 2, this->data.flags[__xThreadInfo.BankId()].get_ucRed(), TmpT.get_xCell());*/
                                //END DEBUG

                                //TmpT.set_Orient((TmpT.get_xOrient() + E_X) % mNrTileOrientations);
				//NOTE: TrivialUND can arise in three ways:
				//1. For some Mirr, there is more than 1 bonding tile T (TrivialUND raised by GetBondingTile)
				//2. For some T, there is more than one orientation O
				//3. T does not agree between all N
				//Else if | T( Mirr( E_X, N(E_X) ) ) | == 0
				//If | T( Mirr( E_X, N(E_X) ) ) | > 0
				//Raise TrivialUND condition
				//Else If | T( Mirr( E_X, N(E_X) ) ) | == 1
				//if ( T.get_xCell() != mEMPTY_CELL ){ //Check if already tile there ??
				if (TmpT.get_xCell() != mEMPTY_CELL) {
                                        TmpT.set_Orient((TmpT.get_xOrient() + E_X) % mNrTileOrientations);
					if( (TmpT.get_xCell() != T.get_xCell()) && (T.get_xCell() != mEMPTY_CELL) ){
						//Raise TrivialUND!
						this->data.flags[__xThreadInfo.WarpId()].set_TrivialUND();
                                                /*BEGIN DEBUG*/
                                                //this->data.grid.set_xCell( __xThreadInfo, 2, 0, this->data.flags[__xThreadInfo.BankId()].get_ucRed(), 3<<2); 
                                                //this->data.grid.set_xCell( __xThreadInfo, 0, 1, this->data.flags[__xThreadInfo.BankId()].get_ucRed(), TmpT.get_xCell() << 2);
                                                //this->data.grid.set_xCell( __xThreadInfo, 0, 2, this->data.flags[__xThreadInfo.BankId()].get_ucRed(), T.get_xCell() << 2);
                                                /*END DEBUG*/
                                                return;
					}
					T.set_xCell(TmpT.get_xCell());
					//As Bonding Cell is rotated such that bonding edge is facing North,
					//we need to rotate tile T such that bonding edge faces bonding site
					//Note: bonding orientations are handled above (GetBondingTile includes orientation).
					//::Let O(T) be all bonding orientations of T
					//If |O(T)| > 1
					//Else If |O(T)| = 1 --> Check Steric, if not --> Assemble
					//Let T* be T rotated such that E_T*(E_X) == E_T(O(T))
					//T.set_Orient((T.get_xOrient() + E_X) % mNrTileOrientations); //Rotate TmpT instead!
				}
			}
		} //Now we have looked for all neighbours of X and filtered the possible bonding tiles
		if (!this->data.flags[__xThreadInfo.BankId()].get_bUNDCondition() && T.get_xCell() != mEMPTY_CELL) {

			//NOTE: StericUND can arise in two ways:
			//1. T does not agree with tile from previous assembly run
			//2. T does not agree with tile already at X in same run (multiple threads only)
                        xCell TmpT2;
			if (this->data.flags[__xThreadInfo.BankId()].get_ucRed() > 0) {
				TmpT2 = this->data.grid.get_xCell(__xThreadInfo, X.x, X.y, this->data.flags[__xThreadInfo.BankId()].get_ucRed() - 1);
				if (TmpT2.get_xCell() != T.get_xCell()) { //We have detected steric non-determinism!
					this->data.flags[__xThreadInfo.BankId()].set_StericUND(); //TEST
                                        /*START DEBUG*/
                                        //this->data.grid.set_xCell( __xThreadInfo, 0, 0, this->data.flags[__xThreadInfo.BankId()].get_ucRed(), 7<<2); 
                                        /*END DEBUG*/
                                        return;
				}
			}

			//If X is not BorderCell
			//Assemble T* at X
			//Note: set_xCell will return false if BorderCell case!
			if (T.get_xCell() != mEMPTY_CELL) { 
                                //BEGIN DEBUG
                                //DBG_COUNTER++;
                                //bool test_flag=false;
                                //END DEBUG
                               
                                //Now: Assemble tile!
				if (!this->data.grid.set_xCell(	__xThreadInfo, X.x, X.y, this->data.flags[__xThreadInfo.BankId()].get_ucRed(), T.get_xCell())) {
					this->data.flags[__xThreadInfo.BankId()].set_UnboundUND(); //TEST
                                        return;
                                        /*START DEBUG*/
                                        //this->data.grid.set_xCell( __xThreadInfo, 1, 0, this->data.flags[__xThreadInfo.BankId()].get_ucRed(), 1<<2); 
                                        //this->data.grid.set_xCell( __xThreadInfo, 0, 1, this->data.flags[__xThreadInfo.BankId()].get_ucRed(), T.get_xCell());
                                        /*END DEBUG*/
                                        //return; //TEST
                                        //test_flag=true;
				}
                                //if(!test_flag){ //DEBUG
				xFourPermutation TmpAddPerm((int) (hiprand_uniform(&this->data.states[__xThreadInfo.BankId()]) * 24.0f));
				unsigned char index2; //Buffer
				while (TmpAddPerm.bNotTraversed()) {
					index2 = TmpAddPerm.ucWalk();
					//For all n(E_X)
					N = this->data.grid.xGetNeighbourCell(__xThreadInfo, X.x, X.y, this->data.flags[__xThreadInfo.BankId()].get_ucRed(), (unsigned char) index2);
					//::Let n(E_X) be empty neighbour cells (i.e. no tile and not on movelist already).
                                        //if(!test_flag){ //DEBUG
					if (N.get_xCell() == mEMPTY_CELL) {
                                                //if(!test_flag){ //DEBUG
                                               
						this->data.movelist.bPush(__xThreadInfo, this->data.grid.xGetNeighbourCellCoords(X.x, X.y, (unsigned char) index2)); 
                                                // } else { //DEBUG
                                                //this->data.grid.set_xCell( __xThreadInfo, 0, 0, this->data.flags[__xThreadInfo.BankId()].get_ucRed(), this->data.grid.xGetNeighbourCellCoords(X.x, X.y, (unsigned char) index2).x);
                                                //this->data.grid.set_xCell( __xThreadInfo, 0, 1, this->data.flags[__xThreadInfo.BankId()].get_ucRed(), this->data.grid.xGetNeighbourCellCoords(X.x, X.y, (unsigned char) index2).y); 
                                                //return;
                                                //} //DEBUG
                                                this->data.grid.set_xCell( __xThreadInfo, this->data.grid.xGetNeighbourCellCoords(X.x, X.y, (unsigned char) (index2)).x, this->data.grid.xGetNeighbourCellCoords(X.x, X.y, (unsigned char) (index2)).y, this->data.flags[__xThreadInfo.BankId()].get_ucRed(), mEMPTY_CELL_ML);
                                                //this->data.grid.set_xCell( __xThreadInfo, 0, index2, this->data.flags[__xThreadInfo.BankId()].get_ucRed(), 99);
					}//}
				}
			} 
		} else { //Remove movelist marking from grid
   	               this->data.grid.set_xCell( __xThreadInfo, X.x, X.y, this->data.flags[__xThreadInfo.BankId()].get_ucRed(), mEMPTY_CELL);
                }

	}
}

__device__ unsigned char xEdgeSort::GetBondingTile(xThreadInfo __xThreadInfo,
                short __sEdgeId, hiprandState *__xCurandState,
                xAssemblyFlags *__xAssemblyFlags) {
        //Takes: Edge Type to which the tile should bond, FitFlags which will be set according to UND conditions
        //Returns: Cell of Bonding Tile type which is rotated such that the bonding tile is facing NORTH (0),
        //If nothing bonds, will return mEMPTY_CELL instead.
        if (this->get_xLength(__xThreadInfo, __sEdgeId) == 1) {
                xCell TmpCell;
                unsigned char TmpBondBuffer = GetBondingTileOrientation(__xThreadInfo,
                                __sEdgeId, 0, __xAssemblyFlags);
                if(TmpBondBuffer == mEMPTY_CELL) return mEMPTY_CELL;
                TmpCell.set_xCell(4 - TmpBondBuffer);
                TmpCell.set_Type(this->get_xData(__xThreadInfo, __sEdgeId, 0,
                                TmpBondBuffer)); //TEST (0 anstelle TmpCell.get_xOrient()) b-fore
                return TmpCell.get_xCell();
        } else if (this->get_xLength(__xThreadInfo, __sEdgeId) == 0) {
                return mEMPTY_CELL;
        } else {
                __xAssemblyFlags->set_TrivialUND();
                return mEMPTY_CELL;
        }
}

__device__ unsigned char xEdgeSort::GetBondingTileOrientation(xThreadInfo __xThreadInfo, unsigned char __ucEdgeId, unsigned char __ucTileId, xAssemblyFlags *__xAssemblyFlags) {
	unsigned char TmpCounter = 0, TmpTile, TmpOrient = mEMPTY_CELL;
	for (int i = 0; i < mNrTileOrientations; i++) {
		TmpTile = this->get_xData(__xThreadInfo, __ucEdgeId, __ucTileId, i);
		if (TmpTile != mEMPTY_CELL) {
			TmpOrient = i;
			TmpCounter++;
			if (TmpCounter >= 2) {
				__xAssemblyFlags->set_TrivialUND();
                                TmpOrient = mEMPTY_CELL;
				break;
			}
		}
	}
	return TmpOrient; //should never be mEMPTY_CELL!
	//Returns edge-id of neighbouring tile that bonds
}

__device__ unsigned char xEdgeSort::get_xData(xThreadInfo __xThreadInfo, unsigned char __ucEdgeId, unsigned char __ucTileId, unsigned char __ucOrientation) {
	return this->data.multi_d[__ucEdgeId][__ucTileId][__ucOrientation][__xThreadInfo.BankId()];
}

__device__ bool xAssembly::Assemble_InPlace(xThreadInfo __xThreadInfo,	xGenomeSet *__xGenomeSet) {
        return true;
}

}

__global__ void TestAssemblyKernel(unsigned char *g_ucGenomes, float *g_ucFitnessValues, unsigned char *g_ucGrids, hiprandState *states)
{
    __shared__ xGenomeSet s_xGenomeSet;
    //__shared__ xEdgeSort s_xEdgeSort;
    __shared__ xAssembly s_xAssembly;
    s_xAssembly.data.states = states;
    xThreadInfo r_xThreadInfo(threadIdx.x, threadIdx.y, blockIdx.x, blockIdx.y);
    s_xGenomeSet.CopyFromGlobal(r_xThreadInfo, g_ucGenomes);
    //s_xEdgeSort.Initialise(r_xThreadInfo, &s_xGenomeSet, -1);
    s_xAssembly.Assemble(r_xThreadInfo, &s_xGenomeSet);
    for(int i=0;i<4;i++){
        //s_xGenomeSet.data.multi_d[r_xThreadInfo.BankId()].data.one_d[i] = s_xEdgeSort.length.multi_d[i][r_xThreadInfo.BankId()];
        //s_xGenomeSet.data.multi_d[r_xThreadInfo.BankId()].data.one_d[i] = s_xEdgeSort.data.multi_d[6][0][i][r_xThreadInfo.BankId()];
        //s_xGenomeSet.data.multi_d[r_xThreadInfo.BankId()].data.one_d[i] = tex2D(t_ucInteractionMatrix, i, 1);
    }
    s_xGenomeSet.CopyToGlobal(r_xThreadInfo, g_ucGenomes); 
    for(int i=0;i<m_fit_DimGridY;i++){
	 for(int j=0;j<m_fit_DimGridX;j++){
             xCell TMP = s_xAssembly.data.grid.get_xCell(r_xThreadInfo, i, j, 0);
             g_ucGrids[r_xThreadInfo.BankId()*m_fit_DimGridX*m_fit_DimGridY + j*m_fit_DimGridX + i] = s_xAssembly.data.grid.get_xCell(r_xThreadInfo, i, j, 0).get_xType();
         }
    }
}


    //end include fitness.cu

    //start include sorting.inc.cu
__global__ void SortingKernel(float *g_fFFValues, float *g_fFFPartialSums){
    __shared__ float s_fFFValues[m_sorting_NR_THREADS_PER_BLOCK];
    
    unsigned int r_uiGlobalThreadOffset = mBLOCK_ID * m_sorting_NR_THREADS_PER_BLOCK + mTHREAD_ID;
    
    if(r_uiGlobalThreadOffset < mNrGenomes){
        s_fFFValues[mTHREAD_ID] = g_fFFValues[r_uiGlobalThreadOffset];
    } else {
        s_fFFValues[mTHREAD_ID] = 0;
    }
    
    __syncthreads();
        
    int i = m_sorting_NR_THREADS_PER_BLOCK / 2; //Do a reduction of fitness values 
    //REQUIRE: m_sorting_NR_THREADS_PER_BLOCK is power of 2
    while(i != 0){
        if(mTHREAD_ID < i){                
            s_fFFValues[mTHREAD_ID] += s_fFFValues[mTHREAD_ID + i];
        } 
        __syncthreads();
        i /= 2;
    }        
    
    __syncthreads();
    if(mTHREAD_ID == 0){
        g_fFFPartialSums[mBLOCK_ID] =  s_fFFValues[0];
    }

    /*if(mTHREAD_ID == 255){
        g_fFFPartialSums[mBLOCK_ID] = m_sorting_NR_THREADS_PER_BLOCK;
    }*/
    return;
}
    //end include sorting.inc.cu

    //start include ga_utils.inc.cu
/*EMPTY*/
/*TESTED*/
__forceinline__ __device__ usGLOBAL1D ga_xGlobalThreadAnchor(ucTYPELENGTH __typelength){ // This is 1D index 0 of a thread in any block in global memory of type of length 
    return  ( (int) __typelength * (mBLOCK_ID * m_ga_THREAD_DIM_X * m_ga_THREAD_DIM_Y + mTHREAD_ID_X * m_ga_THREAD_DIM_Y + mTHREAD_ID_Y ) );
}

/*TESTED*/
__forceinline__ __device__ usSHARED1D ga_xSharedThreadAnchor(ucTYPELENGTH __typelength){
    return __typelength*(mTHREAD_ID_X * m_ga_THREAD_DIM_Y + mTHREAD_ID_Y);
}
    //end include ga_utils.inc.cu   

    //start include ga.inc.cu
extern "C" __global__ void GAKernel(unsigned char *g_ucGenomes,
                                    float *g_fFFValues,
                                    unsigned char *g_ucAssembledGrids,
                                    hiprandState *g_xCurandStates){
    __shared__ unsigned char s_ucGenome[m_ga_NR_THREADS_PER_BLOCK][mByteLengthGenome];    

    int gen_buf;
    float result=0;

    int debug_code = -1;// threadIdx.x + blockIdx.x* m_ga_NR_THREADS_PER_BLOCK; //Test
    if(debug_code==0) printf("Block CODE: %d \n", m_ga_NR_THREADS_PER_BLOCK * mBLOCK_ID + mTHREAD_ID);

    //i.e. exit thread execution immediately if thread is a surplus one
    if ( m_ga_NR_THREADS_PER_BLOCK * mBLOCK_ID + mTHREAD_ID < mNrGenomes){

    //Initialisation: Load Fitness Function Value from global memory
    
    //__syncthreads();
    unsigned int r_uiCutoffIndex = ga_uiSelectionRouletteWheel(&g_xCurandStates[mTHREAD_ID]); //TEST
    if(debug_code==0) printf("CutoffIndex: %d \n", r_uiCutoffIndex);

    //NOTE: FIXED FOR ASEXUAL REPRODUCTION!
    //Note: Initialisation could be done better by accessing global memory coalescently
    if(debug_code==0) printf("GENOME:\n");
    for(int i=0;i<mByteLengthGenome;i++){ //Initialisation: Load Genome from global memory
        s_ucGenome[mTHREAD_ID][i] = g_ucGenomes[ r_uiCutoffIndex * mByteLengthGenome + i ];
//g_ucGenomes[ blockIdx.x * m_ga_NR_THREADS_PER_BLOCK * (mByteLengthGenome) + threadIdx.x * mByteLengthGenome + i ];//ga_xGlobalThreadAnchor(mByteLengthGenome)];
	if(debug_code==0) printf("[%d]=%d,", i, s_ucGenome[mTHREAD_ID][i]);
        //g_ucGenomes[ r_uiCutoffIndex * mByteLengthGenome + i ];// blockIdx.x * m_ga_NR_THREADS_PER_BLOCK * (mByteLengthGenome) + threadIdx.x * mByteLengthGenome + i ];//ga_xGlobalThreadAnchor(mByteLengthGenome)];
    }

    if(debug_code==0) printf("\n");

    /*#ifdef WITH_MIXED_CROSSOVER //I.e. if we have a mixture of single-point and uniform crossover
    float r_fCrossoverSwitch = hiprand_uniform(&g_xCurandStates[mTHREAD_ID]);
    if(r_fCrossoverSwitch >= mUniformCrossoverProbability){
        ga_CrossoverUniform(s_ucGenome, g_ucGenomes, r_uiCutoffIndex, &g_xCurandStates[mTHREAD_ID]);    
    } else {
        ga_CrossoverSinglePoint(s_ucGenome, g_ucGenomes, r_uiCutoffIndex, &g_xCurandStates[mTHREAD_ID]);    
    }
    #else
        #ifdef WITH_UNIFORM_CROSSOVER  
            ga_CrossoverUniform(s_ucGenome, g_ucGenomes, r_uiCutoffIndex, &g_xCurandStates[mTHREAD_ID]);    
        #else
            #ifdef WITH_SINGLE_POINT_CROSSOVER
            ga_CrossoverSinglePoint(s_ucGenome, g_ucGenomes, r_uiCutoffIndex, &g_xCurandStates[mTHREAD_ID]);  //TEST
            #endif
        #endif
    #endif*/
    
    
    
    #ifdef WITH_SUREFIRE_MUTATION
    ga_MutationSurefire(s_ucGenome, m_ga_RateMutation_c, &g_xCurandStates[mTHREAD_ID]);    
    #else
    {
        __shared__ unsigned char s_ucBufGenome[m_ga_NR_THREADS_PER_BLOCK][mByteLengthGenome];
        ga_MutationSophisticated(s_ucGenome, s_ucBufGenome, m_ga_RateMutation_c, &g_xCurandStates[mTHREAD_ID]);    
    }
    #endif
    

    //Evaluate Hamming distance (Fujiama Fitness Function)
    

    
    for(int j=0;j<mByteLengthGenome;j++){ //Copy genome back to global memory
        gen_buf = (int) s_ucGenome[mTHREAD_ID][j];
        gen_buf = gen_buf - ((gen_buf >> 1) & 0x55555555);
        gen_buf = (gen_buf & 0x33333333) + ((gen_buf >> 2) & 0x33333333);
        gen_buf = (((gen_buf + (gen_buf >> 4)) & 0x0F0F0F0F) * 0x01010101) >> 24;
        result += gen_buf;
    }
    }
    
    __syncthreads();
    if ( m_ga_NR_THREADS_PER_BLOCK * mBLOCK_ID + mTHREAD_ID < mNrGenomes){

        g_fFFValues[ m_ga_NR_THREADS_PER_BLOCK * blockIdx.x + threadIdx.x] = result;//(float) result; //CRASHES HERE!
    //g_fFFValues[0] = 1; //CRASHES HERE!
    /*
    if(i == 32 ){
        g_fFFValues[512] = 1.0; //set breaking condition
    }*/
    //return;   
    
    }
    __syncthreads();    
    for(int i=0;i<mByteLengthGenome;i++){ //Copy genome back to global memory
        //g_ucGenomes[ga_xGlobalThreadAnchor(mByteLengthGenome)+i] = s_ucGenome[ga_xSharedThreadAnchor(mByteLengthGenome)][i];
        g_ucGenomes[ blockIdx.x * m_ga_NR_THREADS_PER_BLOCK * (mByteLengthGenome) + threadIdx.x * mByteLengthGenome + i] = s_ucGenome[mTHREAD_ID][i];
    }

}
    //end include ga.inc.cu    
}