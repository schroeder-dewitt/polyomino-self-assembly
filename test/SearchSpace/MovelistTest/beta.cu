#include "hip/hip_runtime.h"
//#include <stdio.h>

//#define m_fit_LengthMovelist 20
//#define mEMPTY_CELL 255

{# {% include "header_inc.cuh" %} #}
{# {% include "fit_header_inc.cuh" %} #}

extern "C++"{

template<class T>
struct xLifoList {
        struct {
                signed short pos; //Current position of top element (-1...max_length-1)
        } data;

        __device__ bool bPush(xThreadInfo __xThreadInfo, T __xEntry, T (&__xStorage)[m_fit_LengthMovelist][mWarpSize], unsigned short __uiMaxLength);
	__device__ T xPop(xThreadInfo __xThreadInfo, T (&__xStorage)[m_fit_LengthMovelist][mWarpSize]);
        __device__ short get_sPos();
        __device__ short set_sPos(short __sPos);
};

template<class T>
__device__ bool xLifoList<T>::bPush(xThreadInfo __xThreadInfo, T __xEntry, T (&__xStorage)[m_fit_LengthMovelist][mWarpSize], unsigned short __uiMaxLength) {
        if (this->data.pos < __uiMaxLength) {
                __xStorage[this->data.pos][__xThreadInfo.BankId()] = __xEntry;
                this->data.pos++;
                return true;
        } else {
                return false;
        }
}

template<class T>
__device__ T xLifoList<T>::xPop(xThreadInfo __xThreadInfo, T (&__xStorage)[m_fit_LengthMovelist][mWarpSize]) {
        if (this->data.pos <= 0) { //FXD
                //NOTE: ADAPTED FOR CUDA VECTORTYPES ONLY!
                T buf;
                buf.x = mEMPTY_CELL;
                buf.y = mEMPTY_CELL;
                return buf;
        } else {
                this->data.pos--;
                return __xStorage[this->data.pos][__xThreadInfo.BankId()];
        }
}

template<class T>
__device__ short xLifoList<T>::get_sPos() {
        return this->data.pos;
}

template<class T>
__device__ short xLifoList<T>::set_sPos(short __sPos){
        this->data.pos = __sPos;
        return 0;
}

template<class T>
struct xMoveList {
        struct {
                T multi_d[m_fit_LengthMovelist][mWarpSize];
        } storage;
        struct {
                xLifoList<T> multi_d[mWarpSize];
        } list;

        __device__ void Initialise(xThreadInfo __xThreadInfo);
        __device__ bool bPush(xThreadInfo __xThreadInfo, T __xEntry);
        __device__ T xPop(xThreadInfo __xThreadInfo);
        __device__ short get_sPos(xThreadInfo __xThreadInfo);
        __device__ short set_sPos(xThreadInfo __xThreadInfo, short __sPos);
};

template<class T>
__device__ bool xMoveList<T>::bPush(xThreadInfo __xThreadInfo, T __xEntry) {
        this->list.multi_d[__xThreadInfo.BankId()].bPush(__xThreadInfo, __xEntry, this->storage.multi_d, {{ fit_length_movelist }});
        return true;
}

template<class T>
__device__ void xMoveList<T>::Initialise(xThreadInfo __xThreadInfo) {
        this->list.multi_d[__xThreadInfo.BankId()].data.pos = 0;
}

template<class T>
__device__ T xMoveList<T>::xPop(xThreadInfo __xThreadInfo) {
	return this->list.multi_d[__xThreadInfo.BankId()].xPop(__xThreadInfo, this->storage.multi_d);
}

template<class T>
__device__ short xMoveList<T>::get_sPos(xThreadInfo __xThreadInfo) {
        return this->list.multi_d[__xThreadInfo.BankId()].get_sPos();
}

template<class T>
__device__ short xMoveList<T>::set_sPos(xThreadInfo __xThreadInfo, short __sPos){
        return this->list.multi_d[__xThreadInfo.BankId()].set_sPos(__sPos);
}

}

//MOVELIST TEST KERNEL
__global__ void TestMovelistKernel(unsigned char *dest)
{
    /*__shared__ xGenomeSet Tmp;
    xThreadInfo Tmpa(threadIdx.x, threadIdx.y, blockIdx.x, blockIdx.y);
    Tmp.CopyFromGlobal(Tmpa, dest);   
    for(int i=0;i<4;i++){
        Tmp.data.multi_d[Tmpa.BankId()].data.one_d[i] = Tmp.get_xEdgeType(Tmpa, i, 1);
    }
    Tmp.CopyToGlobal(Tmpa, dest);*/

    xThreadInfo __xThreadInfo(threadIdx.x, threadIdx.y, blockIdx.x, blockIdx.y); 
    xMoveList<uchar2> s_xMovelist;
    s_xMovelist.Initialise(__xThreadInfo);
    s_xMovelist.bPush(__xThreadInfo, make_uchar2(9,9));
    s_xMovelist.bPush(__xThreadInfo, make_uchar2(6,6));
    for(int i=0;i<2;i++){
        //dest[__xThreadInfo.BankId()*2 + i] = s_xMovelist.storage.multi_d[0][__xThreadInfo.BankId()].x;//s_xMovelist.xPop(__xThreadInfo).x;
        //dest[__xThreadInfo.BankId()*(s_xMovelist.get_sPos(__xThreadInfo)+1) + i] = s_xMovelist.xPop(__xThreadInfo).x;
        //dest[__xThreadInfo.BankId()*(s_xMovelist.get_sPos(__xThreadInfo)+1) + i] = ;
    }
}
