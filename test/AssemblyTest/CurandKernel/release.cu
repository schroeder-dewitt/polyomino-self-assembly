#include "hip/hip_runtime.h"
extern "C++"{
#include <hiprand/hiprand_kernel.h>
}

#define m_curand_NR_THREADS_PER_BLOCK {{ curand_nr_threads_per_block }}
#define m_curand_DimBlockX {{ curand_dim_block_x }}

extern "C" __global__ void CurandInitKernel(hiprandState *state)
{
    int id = m_curand_NR_THREADS_PER_BLOCK * (blockIdx.y * m_curand_DimBlockX + blockIdx.x) + threadIdx.x;
    //Each thread gets same seed, a different sequence number, no offset 
    hiprand_init(1234, id, 0, &state[id]);
    //Solve via offset: http://forums.nvidia.com/index.php?showtopic=185740
    //hiprand_init((5364<<20)+id, 0, 0, &state[id]); //Might lead to collisions - or not...
}

