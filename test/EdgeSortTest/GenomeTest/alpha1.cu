#include "hip/hip_runtime.h"
#include <stdio.h>

{% include "header_inc.cuh" %}
{% include "fit_header_inc.cuh" %}

struct xThreadInfo {
        ushort4 data;
        
        __device__  xThreadInfo(unsigned short __usThreadIdX, unsigned short __usThreadIdY, unsigned short __usBlockIdX, unsigned short __usBlockIdY);
        __device__ unsigned short WarpId(void);
        __device__ unsigned short BankId(void);
        __device__ unsigned short FlatThreadId(void);
        __device__ unsigned short FlatBlockId(void);
        __device__ unsigned short GlobId(unsigned short __usTypeLength);
        __device__ void __DEBUG_CALL(void);
};

__device__ xThreadInfo::xThreadInfo(unsigned short __usThreadIdX, unsigned short __usThreadIdY, unsigned short __usBlockIdX, unsigned short __usBlockIdY) {
        this->data.z = threadIdx.y * m_fit_DimThreadX + threadIdx.x; //Flat Thread ID
        this->data.x = this->data.z % mWarpSize; //BankID
        this->data.y = (this->data.z - this->data.x) / mWarpSize; //WarpID
        this->data.w = blockIdx.y * m_fit_DimBlockX + blockIdx.x; //Flat Block ID
}

__device__ unsigned short xThreadInfo::WarpId(void) {
        return this->data.y;
}
__device__ unsigned short xThreadInfo::BankId(void) {
        return this->data.x;
}
__device__ unsigned short xThreadInfo::FlatThreadId(void) {
        return this->data.z;
}
__device__ unsigned short xThreadInfo::FlatBlockId(void) {
        return this->data.w;
}

__device__ unsigned short xThreadInfo::GlobId(unsigned short __usTypeLength) {
        return (this->data.w * m_fit_DimThreadX * m_fit_DimThreadY + this->data.z) * __usTypeLength;
}

struct xGenome {
        union{
            unsigned char one_d[mAlignedByteLengthGenome];
        } data;

        __device__ void CopyFromGlobal(xThreadInfo __xThreadInfo, unsigned char *__g_ucGenomeSet);
	__device__ void CopyToGlobal(xThreadInfo __xThreadInfo, unsigned char *__g_ucGenomeSet);
        __device__ unsigned char get_xEdgeType(xThreadInfo *__xThreadInfo, unsigned char __ucTileId, unsigned char __ucEdgeId);
        __device__ void set_EdgeType(xThreadInfo *__xThreadInfo, unsigned char __ucTileId, unsigned char __ucEdgeId, unsigned char __ucVal);
};

struct xGenomeSet {
        union{        
            xGenome multi_d[mWarpSize];        
            unsigned char one_d[mWarpSize*sizeof(xGenome)];
        } data;

        __device__ void CopyFromGlobal(xThreadInfo __xThreadInfo, unsigned char *__g_ucGenomeSet);
        __device__ void CopyToGlobal(xThreadInfo __xThreadInfo, unsigned char *__g_ucGenomeSet);
        __device__ unsigned char get_xEdgeType(xThreadInfo *__xThreadInfo, unsigned char __ucTileId, unsigned char __ucEdgeId);
        __device__ unsigned char set_EdgeType(xThreadInfo *__xThreadInfo, unsigned char __ucTileId, unsigned char __ucEdgeId, unsigned char __ucVal);
        __device__ void print(xThreadInfo *__xThreadInfo);
};

__device__ void xGenomeSet::CopyFromGlobal(xThreadInfo __xThreadInfo, unsigned char *__g_ucGenomes) {
        this->data.multi_d[__xThreadInfo.BankId()].CopyFromGlobal(__xThreadInfo, __g_ucGenomes);
}

__device__ void xGenome::CopyFromGlobal(xThreadInfo __xThreadInfo, unsigned char *__g_ucGenomes) {
        for (int i = 0; i < mAlignedByteLengthGenome; i ++) {
             this->data.one_d[i] = __g_ucGenomes[__xThreadInfo.GlobId(sizeof(xGenome)) + i];
        }
}

__device__ void xGenome::CopyToGlobal(xThreadInfo __xThreadInfo, unsigned char *__g_ucGenomes) {
        for (int i = 0; i < mAlignedByteLengthGenome; i += 1) {
                //(*reinterpret_cast<int*> (&this->data.one_d[i])) = (*reinterpret_cast<int*> (&__g_ucGenomeSet[__xThreadInfo->GlobId(sizeof(xGenome)) + i]));
                __g_ucGenomes[__xThreadInfo.GlobId(sizeof(xGenome)) + i] = this->data.one_d[i];
        }
}


__device__ void xGenomeSet::CopyToGlobal(xThreadInfo __xThreadInfo, unsigned char *__g_ucGenomes) {
        this->data.multi_d[__xThreadInfo.BankId()].CopyToGlobal(__xThreadInfo, __g_ucGenomes);
}


//GENOME TEST KERNEL
__global__ void TestGenomeKernel(unsigned char *dest)
{
    __shared__ xGenomeSet Tmp;
    xThreadInfo Tmpa(threadIdx.x, threadIdx.y, blockIdx.x, blockIdx.y);
    //printf("%d |", Tmpa.BankId());
    Tmp.CopyFromGlobal(Tmpa, dest);   
    Tmp.data.multi_d[Tmpa.BankId()].data.one_d[0] = Tmpa.BankId();
    Tmp.CopyToGlobal(Tmpa, dest);
    /*for(int i=0;i<mAlignedByteLengthGenome;i++){
        dest[Tmpa.GlobId(1)+i] = Tmp.data.multi_d[Tmpa.BankId()].data.one_d[i]+10;
    }*/
}
